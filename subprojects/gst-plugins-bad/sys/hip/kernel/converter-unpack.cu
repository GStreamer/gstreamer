
#include <hip/hip_runtime.h>
/* GStreamer
 * Copyright (C) 2025 Seungha Yang <seungha@centricular.com>
 *
 * This library is free software; you can redistribute it and/or
 * modify it under the terms of the GNU Library General Public
 * License as published by the Free Software Foundation; either
 * version 2 of the License, or (at your option) any later version.
 *
 * This library is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
 * Library General Public License for more details.
 *
 * You should have received a copy of the GNU Library General Public
 * License along with this library; if not, write to the
 * Free Software Foundation, Inc., 51 Franklin St, Fifth Floor,
 * Boston, MA 02110-1301, USA.
 */

#if defined(__NVCC__) || defined(__HIPCC__)
#ifdef __HIPCC__
#include <hip/hip_runtime.h>
#endif

extern "C" {
__global__ void
GstHipConverterUnpack_RGB_RGBx
(unsigned char *src, unsigned char *dst, int width, int height,
    int src_stride, int dst_stride)
{
  int x_pos = blockIdx.x * blockDim.x + threadIdx.x;
  int y_pos = blockIdx.y * blockDim.y + threadIdx.y;
  if (x_pos < width && y_pos < height) {
    int dst_pos = x_pos * 4 + y_pos * dst_stride;
    int src_pos = x_pos * 3 + y_pos * src_stride;
    dst[dst_pos] = src[src_pos];
    dst[dst_pos + 1] = src[src_pos + 1];
    dst[dst_pos + 2] = src[src_pos + 2];
    dst[dst_pos + 3] = 0xff;
  }
}

__global__ void
GstHipConverterUnpack_RGB10A2_ARGB64
(unsigned char *src, unsigned char *dst, int width, int height,
    int src_stride, int dst_stride)
{
  int x_pos = blockIdx.x * blockDim.x + threadIdx.x;
  int y_pos = blockIdx.y * blockDim.y + threadIdx.y;
  if (x_pos < width && y_pos < height) {
    unsigned short a, r, g, b;
    unsigned int val;
    int dst_pos = x_pos * 8 + y_pos * dst_stride;
    val = *(unsigned int *)&src[x_pos * 4 + y_pos * src_stride];
    a = (val >> 30) & 0x03;
    a = (a << 14) | (a << 12) | (a << 10) | (a << 8) | (a << 6) | (a << 4) | (a << 2) | (a << 0);
    r = (val & 0x3ff);
    r = (r << 6) | (r >> 4);
    g = ((val >> 10) & 0x3ff);
    g = (g << 6) | (g >> 4);
    b = ((val >> 20) & 0x3ff);
    b = (b << 6) | (b >> 4);
    *(unsigned short *) &dst[dst_pos] = a;
    *(unsigned short *) &dst[dst_pos + 2] = r;
    *(unsigned short *) &dst[dst_pos + 4] = g;
    *(unsigned short *) &dst[dst_pos + 6] = b;
  }
}

__global__ void
GstHipConverterUnpack_BGR10A2_ARGB64
(unsigned char *src, unsigned char *dst, int width, int height,
    int src_stride, int dst_stride)
{
  int x_pos = blockIdx.x * blockDim.x + threadIdx.x;
  int y_pos = blockIdx.y * blockDim.y + threadIdx.y;
  if (x_pos < width && y_pos < height) {
    unsigned short a, r, g, b;
    unsigned int val;
    int dst_pos = x_pos * 8 + y_pos * dst_stride;
    val = *(unsigned int *)&src[x_pos * 4 + y_pos * src_stride];
    a = (val >> 30) & 0x03;
    a = (a << 14) | (a << 12) | (a << 10) | (a << 8) | (a << 6) | (a << 4) | (a << 2) | (a << 0);
    b = (val & 0x3ff);
    b = (b << 6) | (b >> 4);
    g = ((val >> 10) & 0x3ff);
    g = (g << 6) | (g >> 4);
    r = ((val >> 20) & 0x3ff);
    r = (r << 6) | (r >> 4);
    *(unsigned short *) &dst[dst_pos] = a;
    *(unsigned short *) &dst[dst_pos + 2] = r;
    *(unsigned short *) &dst[dst_pos + 4] = g;
    *(unsigned short *) &dst[dst_pos + 6] = b;
  }
}
}
#else
static const char ConverterUnpack_str[] =
"extern \"C\" {\n"
"__global__ void\n"
"GstHipConverterUnpack_RGB_RGBx\n"
"(unsigned char *src, unsigned char *dst, int width, int height,\n"
"    int src_stride, int dst_stride)\n"
"{\n"
"  int x_pos = blockIdx.x * blockDim.x + threadIdx.x;\n"
"  int y_pos = blockIdx.y * blockDim.y + threadIdx.y;\n"
"  if (x_pos < width && y_pos < height) {\n"
"    int dst_pos = x_pos * 4 + y_pos * dst_stride;\n"
"    int src_pos = x_pos * 3 + y_pos * src_stride;\n"
"    dst[dst_pos] = src[src_pos];\n"
"    dst[dst_pos + 1] = src[src_pos + 1];\n"
"    dst[dst_pos + 2] = src[src_pos + 2];\n"
"    dst[dst_pos + 3] = 0xff;\n"
"  }\n"
"}\n"
"\n"
"__global__ void\n"
"GstHipConverterUnpack_RGB10A2_ARGB64\n"
"(unsigned char *src, unsigned char *dst, int width, int height,\n"
"    int src_stride, int dst_stride)\n"
"{\n"
"  int x_pos = blockIdx.x * blockDim.x + threadIdx.x;\n"
"  int y_pos = blockIdx.y * blockDim.y + threadIdx.y;\n"
"  if (x_pos < width && y_pos < height) {\n"
"    unsigned short a, r, g, b;\n"
"    unsigned int val;\n"
"    int dst_pos = x_pos * 8 + y_pos * dst_stride;\n"
"    val = *(unsigned int *)&src[x_pos * 4 + y_pos * src_stride];\n"
"    a = (val >> 30) & 0x03;\n"
"    a = (a << 14) | (a << 12) | (a << 10) | (a << 8) | (a << 6) | (a << 4) | (a << 2) | (a << 0);\n"
"    r = (val & 0x3ff);\n"
"    r = (r << 6) | (r >> 4);\n"
"    g = ((val >> 10) & 0x3ff);\n"
"    g = (g << 6) | (g >> 4);\n"
"    b = ((val >> 20) & 0x3ff);\n"
"    b = (b << 6) | (b >> 4);\n"
"    *(unsigned short *) &dst[dst_pos] = a;\n"
"    *(unsigned short *) &dst[dst_pos + 2] = r;\n"
"    *(unsigned short *) &dst[dst_pos + 4] = g;\n"
"    *(unsigned short *) &dst[dst_pos + 6] = b;\n"
"  }\n"
"}\n"
"\n"
"__global__ void\n"
"GstHipConverterUnpack_BGR10A2_ARGB64\n"
"(unsigned char *src, unsigned char *dst, int width, int height,\n"
"    int src_stride, int dst_stride)\n"
"{\n"
"  int x_pos = blockIdx.x * blockDim.x + threadIdx.x;\n"
"  int y_pos = blockIdx.y * blockDim.y + threadIdx.y;\n"
"  if (x_pos < width && y_pos < height) {\n"
"    unsigned short a, r, g, b;\n"
"    unsigned int val;\n"
"    int dst_pos = x_pos * 8 + y_pos * dst_stride;\n"
"    val = *(unsigned int *)&src[x_pos * 4 + y_pos * src_stride];\n"
"    a = (val >> 30) & 0x03;\n"
"    a = (a << 14) | (a << 12) | (a << 10) | (a << 8) | (a << 6) | (a << 4) | (a << 2) | (a << 0);\n"
"    b = (val & 0x3ff);\n"
"    b = (b << 6) | (b >> 4);\n"
"    g = ((val >> 10) & 0x3ff);\n"
"    g = (g << 6) | (g >> 4);\n"
"    r = ((val >> 20) & 0x3ff);\n"
"    r = (r << 6) | (r >> 4);\n"
"    *(unsigned short *) &dst[dst_pos] = a;\n"
"    *(unsigned short *) &dst[dst_pos + 2] = r;\n"
"    *(unsigned short *) &dst[dst_pos + 4] = g;\n"
"    *(unsigned short *) &dst[dst_pos + 6] = b;\n"
"  }\n"
"}\n"
"}\n"
"\n";
#endif