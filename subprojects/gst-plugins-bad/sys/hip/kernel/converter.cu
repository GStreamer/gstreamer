
#include <hip/hip_runtime.h>
/* GStreamer
 * Copyright (C) 2025 Seungha Yang <seungha@centricular.com>
 *
 * This library is free software; you can redistribute it and/or
 * modify it under the terms of the GNU Library General Public
 * License as published by the Free Software Foundation; either
 * version 2 of the License, or (at your option) any later version.
 *
 * This library is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
 * Library General Public License for more details.
 *
 * You should have received a copy of the GNU Library General Public
 * License along with this library; if not, write to the
 * Free Software Foundation, Inc., 51 Franklin St, Fifth Floor,
 * Boston, MA 02110-1301, USA.
 */

#if defined(__NVCC__) || defined(__HIPCC__)
#ifdef __HIPCC__
#include <hip/hip_runtime.h>
#define TextureObject_t hipTextureObject_t
#else
#define TextureObject_t cudaTextureObject_t
#endif

struct ColorMatrix
{
  float CoeffX[3];
  float CoeffY[3];
  float CoeffZ[3];
  float Offset[3];
  float Min[3];
  float Max[3];
};

struct ConstBuffer
{
  ColorMatrix matrix;
  int width;
  int height;
  int left;
  int top;
  int right;
  int bottom;
  int view_width;
  int view_height;
  float border_x;
  float border_y;
  float border_z;
  float border_w;
  int fill_border;
  int video_direction;
  float alpha;
  int do_blend;
  int do_convert;
};

__device__ inline float
dot (const float coeff[3], float3 val)
{
  return coeff[0] * val.x + coeff[1] * val.y + coeff[2] * val.z;
}

__device__ inline float
clamp (float val, float min_val, float max_val)
{
  return max (min_val, min (val, max_val));
}

__device__ inline float3
clamp3 (float3 val, const float min_val[3], const float max_val[3])
{
  return make_float3 (clamp (val.x, min_val[0], max_val[0]),
      clamp (val.y, min_val[1], max_val[2]),
      clamp (val.z, min_val[1], max_val[2]));
}

__device__ inline unsigned char
scale_to_2bits (float val)
{
  return (unsigned short) __float2int_rz (val * 3.0);
}

__device__ inline unsigned char
scale_to_uchar (float val)
{
  return (unsigned char) __float2int_rz (val * 255.0);
}

__device__ inline unsigned short
scale_to_ushort (float val)
{
  return (unsigned short) __float2int_rz (val * 65535.0);
}

__device__ inline unsigned short
scale_to_10bits (float val)
{
  return (unsigned short) __float2int_rz (val * 1023.0);
}

__device__ inline unsigned short
scale_to_12bits (float val)
{
  return (unsigned short) __float2int_rz (val * 4095.0);
}

__device__ inline unsigned char
blend_uchar (unsigned char dst, float src, float src_alpha)
{
  // DstColor' = SrcA * SrcColor + (1 - SrcA) DstColor
  float src_val = src * src_alpha;
  float dst_val = __int2float_rz (dst) / 255.0 * (1.0 - src_alpha);
  return scale_to_uchar(clamp(src_val + dst_val, 0, 1.0));
}

__device__ inline unsigned short
blend_ushort (unsigned short dst, float src, float src_alpha)
{
  // DstColor' = SrcA * SrcColor + (1 - SrcA) DstColor
  float src_val = src * src_alpha;
  float dst_val = __int2float_rz (dst) / 65535.0 * (1.0 - src_alpha);
  return scale_to_ushort(clamp(src_val + dst_val, 0, 1.0));
}

__device__ inline unsigned short
blend_10bits (unsigned short dst, float src, float src_alpha)
{
  // DstColor' = SrcA * SrcColor + (1 - SrcA) DstColor
  float src_val = src * src_alpha;
  float dst_val = __int2float_rz (dst) / 1023.0 * (1.0 - src_alpha);
  return scale_to_10bits(clamp(src_val + dst_val, 0, 1.0));
}

__device__ inline unsigned short
blend_12bits (unsigned short dst, float src, float src_alpha)
{
  // DstColor' = SrcA * SrcColor + (1 - SrcA) DstColor
  float src_val = src * src_alpha;
  float dst_val = __int2float_rz (dst) / 4095.0 * (1.0 - src_alpha);
  return scale_to_12bits(clamp(src_val + dst_val, 0, 1.0));
}

struct IConverter
{
  __device__ virtual float3
  Execute (float3 sample, const ColorMatrix * matrix) = 0;
};

struct ConvertSimple : public IConverter
{
  __device__ float3
  Execute (float3 sample, const ColorMatrix * matrix)
  {
    float3 out;
    out.x = dot (matrix->CoeffX, sample);
    out.y = dot (matrix->CoeffY, sample);
    out.z = dot (matrix->CoeffZ, sample);
    out.x += matrix->Offset[0];
    out.y += matrix->Offset[1];
    out.z += matrix->Offset[2];
    return clamp3 (out, matrix->Min, matrix->Max);
  }
};

struct ISampler
{
  __device__ virtual float4
  Execute (TextureObject_t tex0, TextureObject_t tex1,
      TextureObject_t tex2, TextureObject_t tex3, float x, float y) = 0;
};

struct SampleI420 : public ISampler
{
  __device__ float4
  Execute (TextureObject_t tex0, TextureObject_t tex1,
      TextureObject_t tex2, TextureObject_t tex3, float x, float y)
  {
    float luma = tex2D<float>(tex0, x, y);
    float u = tex2D<float>(tex1, x, y);
    float v = tex2D<float>(tex2, x, y);
    return make_float4 (luma, u, v, 1);
  }
};

struct SampleYV12 : public ISampler
{
  __device__ float4
  Execute (TextureObject_t tex0, TextureObject_t tex1,
      TextureObject_t tex2, TextureObject_t tex3, float x, float y)
  {
    float luma = tex2D<float>(tex0, x, y);
    float u = tex2D<float>(tex2, x, y);
    float v = tex2D<float>(tex1, x, y);
    return make_float4 (luma, u, v, 1);
  }
};

struct SampleI420_10 : public ISampler
{
  __device__ float4
  Execute (TextureObject_t tex0, TextureObject_t tex1,
      TextureObject_t tex2, TextureObject_t tex3, float x, float y)
  {
    float luma = tex2D<float>(tex0, x, y);
    float u = tex2D<float>(tex1, x, y);
    float v = tex2D<float>(tex2, x, y);
    /* (1 << 6) to scale [0, 1.0) range */
    return make_float4 (luma * 64, u * 64, v * 64, 1);
  }
};

struct SampleI420_12 : public ISampler
{
  __device__ float4
  Execute (TextureObject_t tex0, TextureObject_t tex1,
      TextureObject_t tex2, TextureObject_t tex3, float x, float y)
  {
    float luma = tex2D<float>(tex0, x, y);
    float u = tex2D<float>(tex1, x, y);
    float v = tex2D<float>(tex2, x, y);
    /* (1 << 4) to scale [0, 1.0) range */
    return make_float4 (luma * 16, u * 16, v * 16, 1);
  }
};

struct SampleNV12 : public ISampler
{
  __device__ float4
  Execute (TextureObject_t tex0, TextureObject_t tex1,
      TextureObject_t tex2, TextureObject_t tex3, float x, float y)
  {
    float luma = tex2D<float>(tex0, x, y);
    float2 uv = tex2D<float2>(tex1, x, y);
    return make_float4 (luma, uv.x, uv.y, 1);
  }
};

struct SampleNV21 : public ISampler
{
  __device__ float4
  Execute (TextureObject_t tex0, TextureObject_t tex1,
      TextureObject_t tex2, TextureObject_t tex3, float x, float y)
  {
    float luma = tex2D<float>(tex0, x, y);
    float2 vu = tex2D<float2>(tex1, x, y);
    return make_float4 (luma, vu.y, vu.x, 1);
  }
};

struct SampleRGBA : public ISampler
{
  __device__ float4
  Execute (TextureObject_t tex0, TextureObject_t tex1,
      TextureObject_t tex2, TextureObject_t tex3, float x, float y)
  {
    return tex2D<float4>(tex0, x, y);
  }
};

struct SampleBGRA : public ISampler
{
  __device__ float4
  Execute (TextureObject_t tex0, TextureObject_t tex1,
      TextureObject_t tex2, TextureObject_t tex3, float x, float y)
  {
    float4 bgra = tex2D<float4>(tex0, x, y);
    return make_float4 (bgra.z, bgra.y, bgra.x, bgra.w);
  }
};

struct SampleRGBx : public ISampler
{
  __device__ float4
  Execute (TextureObject_t tex0, TextureObject_t tex1,
      TextureObject_t tex2, TextureObject_t tex3, float x, float y)
  {
    float4 rgbx = tex2D<float4>(tex0, x, y);
    rgbx.w = 1;
    return rgbx;
  }
};

struct SampleBGRx : public ISampler
{
  __device__ float4
  Execute (TextureObject_t tex0, TextureObject_t tex1,
      TextureObject_t tex2, TextureObject_t tex3, float x, float y)
  {
    float4 bgrx = tex2D<float4>(tex0, x, y);
    return make_float4 (bgrx.z, bgrx.y, bgrx.x, 1);
  }
};

struct SampleARGB : public ISampler
{
  __device__ float4
  Execute (TextureObject_t tex0, TextureObject_t tex1,
      TextureObject_t tex2, TextureObject_t tex3, float x, float y)
  {
   float4 argb = tex2D<float4>(tex0, x, y);
   return make_float4 (argb.y, argb.z, argb.w, argb.x);
  }
};

struct SampleABGR : public ISampler
{
  __device__ float4
  Execute (TextureObject_t tex0, TextureObject_t tex1,
      TextureObject_t tex2, TextureObject_t tex3, float x, float y)
  {
   float4 abgr = tex2D<float4>(tex0, x, y);
   return make_float4 (abgr.w, abgr.z, abgr.y, abgr.x);
  }
};

struct SampleRGBP : public ISampler
{
  __device__ float4
  Execute (TextureObject_t tex0, TextureObject_t tex1,
      TextureObject_t tex2, TextureObject_t tex3, float x, float y)
  {
    float r = tex2D<float>(tex0, x, y);
    float g = tex2D<float>(tex1, x, y);
    float b = tex2D<float>(tex2, x, y);
    return make_float4 (r, g, b, 1);
  }
};

struct SampleBGRP : public ISampler
{
  __device__ float4
  Execute (TextureObject_t tex0, TextureObject_t tex1,
      TextureObject_t tex2, TextureObject_t tex3, float x, float y)
  {
    float b = tex2D<float>(tex0, x, y);
    float g = tex2D<float>(tex1, x, y);
    float r = tex2D<float>(tex2, x, y);
    return make_float4 (r, g, b, 1);
  }
};

struct SampleGBR : public ISampler
{
  __device__ float4
  Execute (TextureObject_t tex0, TextureObject_t tex1,
      TextureObject_t tex2, TextureObject_t tex3, float x, float y)
  {
    float g = tex2D<float>(tex0, x, y);
    float b = tex2D<float>(tex1, x, y);
    float r = tex2D<float>(tex2, x, y);
    return make_float4 (r, g, b, 1);
  }
};

struct SampleGBR_10 : public ISampler
{
  __device__ float4
  Execute (TextureObject_t tex0, TextureObject_t tex1,
      TextureObject_t tex2, TextureObject_t tex3, float x, float y)
  {
    float g = tex2D<float>(tex0, x, y);
    float b = tex2D<float>(tex1, x, y);
    float r = tex2D<float>(tex2, x, y);
    /* (1 << 6) to scale [0, 1.0) range */
    return make_float4 (r * 64, g * 64, b * 64, 1);
  }
};

struct SampleGBR_12 : public ISampler
{
  __device__ float4
  Execute (TextureObject_t tex0, TextureObject_t tex1,
      TextureObject_t tex2, TextureObject_t tex3, float x, float y)
  {
    float g = tex2D<float>(tex0, x, y);
    float b = tex2D<float>(tex1, x, y);
    float r = tex2D<float>(tex2, x, y);
    /* (1 << 4) to scale [0, 1.0) range */
    return make_float4 (r * 16, g * 16, b * 16, 1);
  }
};

struct SampleGBRA : public ISampler
{
  __device__ float4
  Execute (TextureObject_t tex0, TextureObject_t tex1,
      TextureObject_t tex2, TextureObject_t tex3, float x, float y)
  {
    float g = tex2D<float>(tex0, x, y);
    float b = tex2D<float>(tex1, x, y);
    float r = tex2D<float>(tex2, x, y);
    float a = tex2D<float>(tex3, x, y);
    return make_float4 (r, g, b, a);
  }
};

struct SampleVUYA : public ISampler
{
  __device__ float4
  Execute (TextureObject_t tex0, TextureObject_t tex1,
      TextureObject_t tex2, TextureObject_t tex3, float x, float y)
  {
    float4 vuya = tex2D<float4>(tex0, x, y);
    return make_float4 (vuya.z, vuya.y, vuya.x, vuya.w);
  }
};

struct IOutput
{
  __device__ virtual void
  Write (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,
      unsigned char * dst3, float4 sample, int x, int y, int stride0,
      int stride1) = 0;

  __device__ virtual void
  Blend (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,
        unsigned char * dst3, float4 sample, int x, int y, int stride0,
        int stride1) = 0;
};

struct OutputI420 : public IOutput
{
  __device__ void
  Write (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,
      unsigned char * dst3, float4 sample, int x, int y, int stride0,
      int stride1)
  {
    dst0[x + y * stride0] = scale_to_uchar (sample.x);
    if (x % 2 == 0 && y % 2 == 0) {
      unsigned int pos = x / 2 + (y / 2) * stride1;
      dst1[pos] = scale_to_uchar (sample.y);
      dst2[pos] = scale_to_uchar (sample.z);
    }
  }

  __device__ void
  Blend (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,
      unsigned char * dst3, float4 sample, int x, int y, int stride0,
      int stride1)
  {
    unsigned int pos = x + y * stride0;
    dst0[pos] = blend_uchar (dst0[pos], sample.x, sample.w);
    if (x % 2 == 0 && y % 2 == 0) {
      pos = x / 2 + (y / 2) * stride1;
      dst1[pos] = blend_uchar (dst1[pos], sample.y, sample.w);
      dst2[pos] = blend_uchar (dst2[pos], sample.z, sample.w);
    }
  }
};

struct OutputYV12 : public IOutput
{
  __device__ void
  Write (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,
      unsigned char * dst3, float4 sample, int x, int y, int stride0,
      int stride1)
  {
    dst0[x + y * stride0] = scale_to_uchar (sample.x);
    if (x % 2 == 0 && y % 2 == 0) {
      unsigned int pos = x / 2 + (y / 2) * stride1;
      dst1[pos] = scale_to_uchar (sample.z);
      dst2[pos] = scale_to_uchar (sample.y);
    }
  }

  __device__ void
  Blend (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,
      unsigned char * dst3, float4 sample, int x, int y, int stride0,
      int stride1)
  {
    unsigned int pos = x + y * stride0;
    dst0[pos] = blend_uchar (dst0[pos], sample.x, sample.w);
    if (x % 2 == 0 && y % 2 == 0) {
      pos = x / 2 + (y / 2) * stride1;
      dst1[pos] = blend_uchar (dst1[pos], sample.z, sample.w);
      dst2[pos] = blend_uchar (dst2[pos], sample.y, sample.w);
    }
  }
};

struct OutputNV12 : public IOutput
{
  __device__ void
  Write (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,
      unsigned char * dst3, float4 sample, int x, int y, int stride0,
      int stride1)
  {
    dst0[x + y * stride0] = scale_to_uchar (sample.x);
    if (x % 2 == 0 && y % 2 == 0) {
      unsigned int pos = x + (y / 2) * stride1;
      dst1[pos] = scale_to_uchar (sample.y);
      dst1[pos + 1] = scale_to_uchar (sample.z);
    }
  }

  __device__ void
  Blend (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,
      unsigned char * dst3, float4 sample, int x, int y, int stride0,
      int stride1)
  {
    unsigned int pos = x + y * stride0;
    dst0[pos] = blend_uchar (dst0[pos], sample.x, sample.w);
    if (x % 2 == 0 && y % 2 == 0) {
      pos = x + (y / 2) * stride1;
      dst1[pos] = blend_uchar (dst1[pos], sample.y, sample.w);
      dst1[pos + 1] = blend_uchar (dst1[pos + 1], sample.z, sample.w);
    }
  }
};

struct OutputNV21 : public IOutput
{
  __device__ void
  Write (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,
      unsigned char * dst3, float4 sample, int x, int y, int stride0,
      int stride1)
  {
    dst0[x + y * stride0] = scale_to_uchar (sample.x);
    if (x % 2 == 0 && y % 2 == 0) {
      unsigned int pos = x + (y / 2) * stride1;
      dst1[pos] = scale_to_uchar (sample.z);
      dst1[pos + 1] = scale_to_uchar (sample.y);
    }
  }

  __device__ void
  Blend (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,
      unsigned char * dst3, float4 sample, int x, int y, int stride0,
      int stride1)
  {
    unsigned int pos = x + y * stride0;
    dst0[pos] = blend_uchar (dst0[pos], sample.x, sample.w);
    if (x % 2 == 0 && y % 2 == 0) {
      pos = x + (y / 2) * stride1;
      dst1[pos] = blend_uchar (dst1[pos], sample.z, sample.w);
      dst1[pos + 1] = blend_uchar (dst1[pos + 1], sample.y, sample.w);
    }
  }
};

struct OutputP010 : public IOutput
{
  __device__ void
  Write (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,
      unsigned char * dst3, float4 sample, int x, int y, int stride0,
      int stride1)
  {
    *(unsigned short *) &dst0[x * 2 + y * stride0] = scale_to_ushort (sample.x);
    if (x % 2 == 0 && y % 2 == 0) {
      unsigned int pos = x * 2 + (y / 2) * stride1;
      *(unsigned short *) &dst1[pos] = scale_to_ushort (sample.y);
      *(unsigned short *) &dst1[pos + 2] = scale_to_ushort (sample.z);
    }
  }

  __device__ void
  Blend (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,
      unsigned char * dst3, float4 sample, int x, int y, int stride0,
      int stride1)
  {
    unsigned int pos = x * 2 + y * stride0;
    unsigned short * target = (unsigned short *) &dst0[pos];
    *target = blend_ushort (*target, sample.x, sample.w);
    if (x % 2 == 0 && y % 2 == 0) {
      pos = x * 2 + (y / 2) * stride1;
      target = (unsigned short *) &dst1[pos];
      *target = blend_ushort (*target, sample.y, sample.w);
      target = (unsigned short *) &dst1[pos + 2];
      *target = blend_ushort (*target, sample.z, sample.w);
    }
  }
};

struct OutputI420_10 : public IOutput
{
  __device__ void
  Write (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,
      unsigned char * dst3, float4 sample, int x, int y, int stride0,
      int stride1)
  {
    *(unsigned short *) &dst0[x * 2 + y * stride0] = scale_to_10bits (sample.x);
    if (x % 2 == 0 && y % 2 == 0) {
      unsigned int pos = x + (y / 2) * stride1;
      *(unsigned short *) &dst1[pos] = scale_to_10bits (sample.y);
      *(unsigned short *) &dst2[pos] = scale_to_10bits (sample.z);
    }
  }

  __device__ void
  Blend (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,
      unsigned char * dst3, float4 sample, int x, int y, int stride0,
      int stride1)
  {
    unsigned int pos = x * 2 + y * stride0;
    unsigned short * target = (unsigned short *) &dst0[pos];
    *target = blend_10bits (*target, sample.x, sample.w);
    if (x % 2 == 0 && y % 2 == 0) {
      pos = x * 2 + (y / 2) * stride1;
      target = (unsigned short *) &dst1[pos];
      *target = blend_10bits (*target, sample.y, sample.w);
      target = (unsigned short *) &dst2[pos];
      *target = blend_10bits (*target, sample.z, sample.w);
    }
  }
};

struct OutputI420_12 : public IOutput
{
  __device__ void
  Write (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,
      unsigned char * dst3, float4 sample, int x, int y, int stride0,
      int stride1)
  {
    *(unsigned short *) &dst0[x * 2 + y * stride0] = scale_to_12bits (sample.x);
    if (x % 2 == 0 && y % 2 == 0) {
      unsigned int pos = x + (y / 2) * stride1;
      *(unsigned short *) &dst1[pos] = scale_to_12bits (sample.y);
      *(unsigned short *) &dst2[pos] = scale_to_12bits (sample.z);
    }
  }

  __device__ void
  Blend (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,
      unsigned char * dst3, float4 sample, int x, int y, int stride0,
      int stride1)
  {
    unsigned int pos = x * 2 + y * stride0;
    unsigned short * target = (unsigned short *) &dst0[pos];
    *target = blend_12bits (*target, sample.x, sample.w);
    if (x % 2 == 0 && y % 2 == 0) {
      pos = x * 2 + (y / 2) * stride1;
      target = (unsigned short *) &dst1[pos];
      *target = blend_12bits (*target, sample.y, sample.w);
      target = (unsigned short *) &dst2[pos];
      *target = blend_12bits (*target, sample.z, sample.w);
    }
  }
};

struct OutputY444 : public IOutput
{
  __device__ void
  Write (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,
      unsigned char * dst3, float4 sample, int x, int y, int stride0,
      int stride1)
  {
    int pos = x + y * stride0;
    dst0[pos] = scale_to_uchar (sample.x);
    dst1[pos] = scale_to_uchar (sample.y);
    dst2[pos] = scale_to_uchar (sample.z);
  }

  __device__ void
  Blend (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,
      unsigned char * dst3, float4 sample, int x, int y, int stride0,
      int stride1)
  {
    int pos = x + y * stride0;
    dst0[pos] = blend_uchar (dst0[pos], sample.x, sample.w);
    dst1[pos] = blend_uchar (dst1[pos], sample.y, sample.w);
    dst2[pos] = blend_uchar (dst2[pos], sample.z, sample.w);
  }
};

struct OutputY444_10 : public IOutput
{
  __device__ void
  Write (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,
      unsigned char * dst3, float4 sample, int x, int y, int stride0,
      int stride1)
  {
    int pos = x * 2 + y * stride0;
    *(unsigned short *) &dst0[pos] = scale_to_10bits (sample.x);
    *(unsigned short *) &dst1[pos] = scale_to_10bits (sample.y);
    *(unsigned short *) &dst2[pos] = scale_to_10bits (sample.z);
  }

  __device__ void
  Blend (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,
      unsigned char * dst3, float4 sample, int x, int y, int stride0,
      int stride1)
  {
    int pos = x * 2 + y * stride0;
    unsigned short * target = (unsigned short *) &dst0[pos];
    *target = blend_10bits (*target, sample.x, sample.w);
    target = (unsigned short *) &dst1[pos];
    *target = blend_10bits (*target, sample.y, sample.w);
    target = (unsigned short *) &dst2[pos];
    *target = blend_10bits (*target, sample.z, sample.w);
  }
};

struct OutputY444_12 : public IOutput
{
  __device__ void
  Write (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,
      unsigned char * dst3, float4 sample, int x, int y, int stride0,
      int stride1)
  {
    int pos = x * 2 + y * stride0;
    *(unsigned short *) &dst0[pos] = scale_to_12bits (sample.x);
    *(unsigned short *) &dst1[pos] = scale_to_12bits (sample.y);
    *(unsigned short *) &dst2[pos] = scale_to_12bits (sample.z);
  }

  __device__ void
  Blend (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,
      unsigned char * dst3, float4 sample, int x, int y, int stride0,
      int stride1)
  {
    int pos = x * 2 + y * stride0;
    unsigned short * target = (unsigned short *) &dst0[pos];
    *target = blend_12bits (*target, sample.x, sample.w);
    target = (unsigned short *) &dst1[pos];
    *target = blend_12bits (*target, sample.y, sample.w);
    target = (unsigned short *) &dst2[pos];
    *target = blend_12bits (*target, sample.z, sample.w);
  }
};

struct OutputY444_16 : public IOutput
{
  __device__ void
  Write (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,
      unsigned char * dst3, float4 sample, int x, int y, int stride0,
      int stride1)
  {
    int pos = x * 2 + y * stride0;
    *(unsigned short *) &dst0[pos] = scale_to_ushort (sample.x);
    *(unsigned short *) &dst1[pos] = scale_to_ushort (sample.y);
    *(unsigned short *) &dst2[pos] = scale_to_ushort (sample.z);
  }

  __device__ void
  Blend (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,
      unsigned char * dst3, float4 sample, int x, int y, int stride0,
      int stride1)
  {
    int pos = x * 2 + y * stride0;
    unsigned short * target = (unsigned short *) &dst0[pos];
    *target = blend_ushort (*target, sample.x, sample.w);
    target = (unsigned short *) &dst1[pos];
    *target = blend_ushort (*target, sample.y, sample.w);
    target = (unsigned short *) &dst2[pos];
    *target = blend_ushort (*target, sample.z, sample.w);
  }
};

struct OutputRGBA : public IOutput
{
  __device__ void
  Write (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,
      unsigned char * dst3, float4 sample, int x, int y, int stride0,
      int stride1)
  {
    int pos = x * 4 + y * stride0;
    dst0[pos] = scale_to_uchar (sample.x);
    dst0[pos + 1] = scale_to_uchar (sample.y);
    dst0[pos + 2] = scale_to_uchar (sample.z);
    dst0[pos + 3] = scale_to_uchar (sample.w);
  }

  __device__ void
  Blend (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,
      unsigned char * dst3, float4 sample, int x, int y, int stride0,
      int stride1)
  {
    int pos = x * 4 + y * stride0;
    dst0[pos] = blend_uchar (dst0[pos], sample.x, sample.w);
    dst0[pos + 1] = blend_uchar (dst0[pos + 1], sample.y, sample.w);
    dst0[pos + 2] = blend_uchar (dst0[pos + 2], sample.z, sample.w);
    dst0[pos + 3] = blend_uchar (dst0[pos + 3], 1.0, sample.w);
  }
};

struct OutputRGBx : public IOutput
{
  __device__ void
  Write (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,
      unsigned char * dst3, float4 sample, int x, int y, int stride0,
      int stride1)
  {
    int pos = x * 4 + y * stride0;
    dst0[pos] = scale_to_uchar (sample.x);
    dst0[pos + 1] = scale_to_uchar (sample.y);
    dst0[pos + 2] = scale_to_uchar (sample.z);
    dst0[pos + 3] = 255;
  }

  __device__ void
  Blend (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,
      unsigned char * dst3, float4 sample, int x, int y, int stride0,
      int stride1)
  {
    int pos = x * 4 + y * stride0;
    dst0[pos] = blend_uchar (dst0[pos], sample.x, sample.w);
    dst0[pos + 1] = blend_uchar (dst0[pos + 1], sample.y, sample.w);
    dst0[pos + 2] = blend_uchar (dst0[pos + 2], sample.z, sample.w);
    dst0[pos + 3] = 255;
  }
};

struct OutputBGRA : public IOutput
{
  __device__ void
  Write (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,
      unsigned char * dst3, float4 sample, int x, int y, int stride0,
      int stride1)
  {
    int pos = x * 4 + y * stride0;
    dst0[pos] = scale_to_uchar (sample.z);
    dst0[pos + 1] = scale_to_uchar (sample.y);
    dst0[pos + 2] = scale_to_uchar (sample.x);
    dst0[pos + 3] = scale_to_uchar (sample.w);
  }

  __device__ void
  Blend (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,
      unsigned char * dst3, float4 sample, int x, int y, int stride0,
      int stride1)
  {
    int pos = x * 4 + y * stride0;
    dst0[pos] = blend_uchar (dst0[pos], sample.z, sample.w);
    dst0[pos + 1] = blend_uchar (dst0[pos + 1], sample.y, sample.w);
    dst0[pos + 2] = blend_uchar (dst0[pos + 2], sample.x, sample.w);
    dst0[pos + 3] = blend_uchar (dst0[pos + 3], 1.0, sample.w);
  }
};

struct OutputBGRx : public IOutput
{
  __device__ void
  Write (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,
      unsigned char * dst3, float4 sample, int x, int y, int stride0,
      int stride1)
  {
    int pos = x * 4 + y * stride0;
    dst0[pos] = scale_to_uchar (sample.z);
    dst0[pos + 1] = scale_to_uchar (sample.y);
    dst0[pos + 2] = scale_to_uchar (sample.x);
    dst0[pos + 3] = 255;
  }

  __device__ void
  Blend (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,
      unsigned char * dst3, float4 sample, int x, int y, int stride0,
      int stride1)
  {
    int pos = x * 4 + y * stride0;
    dst0[pos] = blend_uchar (dst0[pos], sample.z, sample.w);
    dst0[pos + 1] = blend_uchar (dst0[pos + 1], sample.y, sample.w);
    dst0[pos + 2] = blend_uchar (dst0[pos + 2], sample.x, sample.w);
    dst0[pos + 3] = 255;
  }
};

struct OutputARGB : public IOutput
{
  __device__ void
  Write (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,
      unsigned char * dst3, float4 sample, int x, int y, int stride0,
      int stride1)
  {
    int pos = x * 4 + y * stride0;
    dst0[pos] = scale_to_uchar (sample.w);
    dst0[pos + 1] = scale_to_uchar (sample.x);
    dst0[pos + 2] = scale_to_uchar (sample.y);
    dst0[pos + 3] = scale_to_uchar (sample.z);
  }

  __device__ void
  Blend (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,
      unsigned char * dst3, float4 sample, int x, int y, int stride0,
      int stride1)
  {
    int pos = x * 4 + y * stride0;
    dst0[pos] = blend_uchar (dst0[pos], 1.0, sample.w);
    dst0[pos + 1] = blend_uchar (dst0[pos + 1], sample.x, sample.w);
    dst0[pos + 2] = blend_uchar (dst0[pos + 2], sample.y, sample.w);
    dst0[pos + 3] = blend_uchar (dst0[pos + 3], sample.z, sample.w);
  }
};

struct OutputABGR : public IOutput
{
  __device__ void
  Write (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,
      unsigned char * dst3, float4 sample, int x, int y, int stride0,
      int stride1)
  {
    int pos = x * 4 + y * stride0;
    dst0[pos] = scale_to_uchar (sample.w);
    dst0[pos + 1] = scale_to_uchar (sample.z);
    dst0[pos + 2] = scale_to_uchar (sample.y);
    dst0[pos + 3] = scale_to_uchar (sample.x);
  }

  __device__ void
  Blend (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,
      unsigned char * dst3, float4 sample, int x, int y, int stride0,
      int stride1)
  {
    int pos = x * 4 + y * stride0;
    dst0[pos] = blend_uchar (dst0[pos], 1.0, sample.w);
    dst0[pos + 1] = blend_uchar (dst0[pos + 1], sample.z, sample.w);
    dst0[pos + 2] = blend_uchar (dst0[pos + 2], sample.y, sample.w);
    dst0[pos + 3] = blend_uchar (dst0[pos + 3], sample.x, sample.w);
  }
};

struct OutputRGB : public IOutput
{
  __device__ void
  Write (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,
      unsigned char * dst3, float4 sample, int x, int y, int stride0,
      int stride1)
  {
    int pos = x * 3 + y * stride0;
    dst0[pos] = scale_to_uchar (sample.x);
    dst0[pos + 1] = scale_to_uchar (sample.y);
    dst0[pos + 2] = scale_to_uchar (sample.z);
  }

  __device__ void
  Blend (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,
      unsigned char * dst3, float4 sample, int x, int y, int stride0,
      int stride1)
  {
    int pos = x * 3 + y * stride0;
    dst0[pos] = blend_uchar (dst0[pos], sample.x, sample.w);
    dst0[pos + 1] = blend_uchar (dst0[pos + 1], sample.y, sample.w);
    dst0[pos + 2] = blend_uchar (dst0[pos + 2], sample.z, sample.w);
  }
};

struct OutputBGR : public IOutput
{
  __device__ void
  Write (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,
      unsigned char * dst3, float4 sample, int x, int y, int stride0,
      int stride1)
  {
    int pos = x * 3 + y * stride0;
    dst0[pos] = scale_to_uchar (sample.z);
    dst0[pos + 1] = scale_to_uchar (sample.y);
    dst0[pos + 2] = scale_to_uchar (sample.x);
  }

  __device__ void
  Blend (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,
      unsigned char * dst3, float4 sample, int x, int y, int stride0,
      int stride1)
  {
    int pos = x * 3 + y * stride0;
    dst0[pos] = blend_uchar (dst0[pos], sample.z, sample.w);
    dst0[pos + 1] = blend_uchar (dst0[pos + 1], sample.y, sample.w);
    dst0[pos + 2] = blend_uchar (dst0[pos + 2], sample.x, sample.w);
  }
};

__device__ inline ushort3
unpack_rgb10a2 (unsigned int val)
{
  unsigned short r, g, b;
  r = (val & 0x3ff);
  r = (r << 6) | (r >> 4);
  g = ((val >> 10) & 0x3ff);
  g = (g << 6) | (g >> 4);
  b = ((val >> 20) & 0x3ff);
  b = (b << 6) | (b >> 4);
  return make_ushort3 (r, g, b);
}

struct OutputRGB10A2 : public IOutput
{
  __device__ void
  Write (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,
      unsigned char * dst3, float4 sample, int x, int y, int stride0,
      int stride1)
  {
    unsigned int alpha = (unsigned int) scale_to_2bits (sample.w);
    unsigned int packed_rgb = alpha << 30;
    packed_rgb |= ((unsigned int) scale_to_10bits (sample.x));
    packed_rgb |= ((unsigned int) scale_to_10bits (sample.y)) << 10;
    packed_rgb |= ((unsigned int) scale_to_10bits (sample.z)) << 20;
    *(unsigned int *) &dst0[x * 4 + y * stride0] = packed_rgb;
  }

  __device__ void
  Blend (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,
      unsigned char * dst3, float4 sample, int x, int y, int stride0,
      int stride1)
  {
    unsigned int * target = (unsigned int *) &dst0[x * 4 + y * stride0];
    ushort3 val = unpack_rgb10a2 (*target);
    unsigned int alpha = (unsigned int) scale_to_2bits (sample.w);
    unsigned int packed_rgb = alpha << 30;
    packed_rgb |= ((unsigned int) blend_10bits (val.x, sample.x, sample.w));
    packed_rgb |= ((unsigned int) blend_10bits (val.y, sample.y, sample.w)) << 10;
    packed_rgb |= ((unsigned int) blend_10bits (val.z, sample.z, sample.w)) << 20;
    *target = packed_rgb;
  }
};

__device__ inline ushort3
unpack_bgr10a2 (unsigned int val)
{
  unsigned short r, g, b;
  b = (val & 0x3ff);
  b = (b << 6) | (b >> 4);
  g = ((val >> 10) & 0x3ff);
  g = (g << 6) | (g >> 4);
  r = ((val >> 20) & 0x3ff);
  r = (r << 6) | (r >> 4);
  return make_ushort3 (r, g, b);
}

struct OutputBGR10A2 : public IOutput
{
  __device__ void
  Write (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,
      unsigned char * dst3, float4 sample, int x, int y, int stride0,
      int stride1)
  {
    unsigned int alpha = (unsigned int) scale_to_2bits (sample.x);
    unsigned int packed_rgb = alpha << 30;
    packed_rgb |= ((unsigned int) scale_to_10bits (sample.x)) << 20;
    packed_rgb |= ((unsigned int) scale_to_10bits (sample.y)) << 10;
    packed_rgb |= ((unsigned int) scale_to_10bits (sample.z));
    *(unsigned int *) &dst0[x * 4 + y * stride0] = packed_rgb;
  }

  __device__ void
  Blend (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,
      unsigned char * dst3, float4 sample, int x, int y, int stride0,
      int stride1)
  {
    unsigned int * target = (unsigned int *) &dst0[x * 4 + y * stride0];
    ushort3 val = unpack_bgr10a2 (*target);
    unsigned int alpha = (unsigned int) scale_to_2bits (sample.w);
    unsigned int packed_rgb = alpha << 30;
    packed_rgb |= ((unsigned int) blend_10bits (val.x, sample.x, sample.w)) << 20;
    packed_rgb |= ((unsigned int) blend_10bits (val.y, sample.y, sample.w)) << 10;
    packed_rgb |= ((unsigned int) blend_10bits (val.z, sample.z, sample.w));
    *target = packed_rgb;
  }
};

struct OutputY42B : public IOutput
{
  __device__ void
  Write (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,
      unsigned char * dst3, float4 sample, int x, int y, int stride0,
      int stride1)
  {
    dst0[x + y * stride0] = scale_to_uchar (sample.x);
    if (x % 2 == 0) {
      unsigned int pos = x / 2 + y * stride1;
      dst1[pos] = scale_to_uchar (sample.y);
      dst2[pos] = scale_to_uchar (sample.z);
    }
  }

  __device__ void
  Blend (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,
      unsigned char * dst3, float4 sample, int x, int y, int stride0,
      int stride1)
  {
    unsigned int pos = x + y * stride0;
    dst0[pos] = blend_uchar (dst0[pos], sample.x, sample.w);
    if (x % 2 == 0) {
      pos = x / 2 + y * stride1;
      dst1[pos] = blend_uchar (dst1[pos], sample.y, sample.w);
      dst2[pos] = blend_uchar (dst2[pos], sample.z, sample.w);
    }
  }
};

struct OutputI422_10 : public IOutput
{
  __device__ void
  Write (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,
      unsigned char * dst3, float4 sample, int x, int y, int stride0,
      int stride1)
  {
    *(unsigned short *) &dst0[x * 2 + y * stride0] = scale_to_10bits (sample.x);
    if (x % 2 == 0) {
      unsigned int pos = x + y * stride1;
      *(unsigned short *) &dst1[pos] = scale_to_10bits (sample.y);
      *(unsigned short *) &dst2[pos] = scale_to_10bits (sample.z);
    }
  }

  __device__ void
  Blend (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,
      unsigned char * dst3, float4 sample, int x, int y, int stride0,
      int stride1)
  {
    unsigned int pos = x * 2 + y * stride0;
    unsigned short * target = (unsigned short *) &dst0[pos];
    *target = blend_10bits (*target, sample.x, sample.w);
    if (x % 2 == 0) {
      pos = x / 2 + y * stride1;
      target = (unsigned short *) &dst1[pos];
      *target = blend_10bits (*target, sample.y, sample.w);
      target = (unsigned short *) &dst2[pos];
      *target = blend_10bits (*target, sample.z, sample.w);
    }
  }
};

struct OutputI422_12 : public IOutput
{
  __device__ void
  Write (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,
      unsigned char * dst3, float4 sample, int x, int y, int stride0,
      int stride1)
  {
    *(unsigned short *) &dst0[x * 2 + y * stride0] = scale_to_12bits (sample.x);
    if (x % 2 == 0) {
      unsigned int pos = x + y * stride1;
      *(unsigned short *) &dst1[pos] = scale_to_12bits (sample.y);
      *(unsigned short *) &dst2[pos] = scale_to_12bits (sample.z);
    }
  }

  __device__ void
  Blend (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,
      unsigned char * dst3, float4 sample, int x, int y, int stride0,
      int stride1)
  {
    unsigned int pos = x * 2 + y * stride0;
    unsigned short * target = (unsigned short *) &dst0[pos];
    *target = blend_12bits (*target, sample.x, sample.w);
    if (x % 2 == 0) {
      pos = x / 2 + y * stride1;
      target = (unsigned short *) &dst1[pos];
      *target = blend_12bits (*target, sample.y, sample.w);
      target = (unsigned short *) &dst2[pos];
      *target = blend_12bits (*target, sample.z, sample.w);
    }
  }
};

struct OutputRGBP : public IOutput
{
  __device__ void
  Write (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,
      unsigned char * dst3, float4 sample, int x, int y, int stride0,
      int stride1)
  {
    int pos = x + y * stride0;
    dst0[pos] = scale_to_uchar (sample.x);
    dst1[pos] = scale_to_uchar (sample.y);
    dst2[pos] = scale_to_uchar (sample.z);
  }

  __device__ void
  Blend (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,
      unsigned char * dst3, float4 sample, int x, int y, int stride0,
      int stride1)
  {
    int pos = x + y * stride0;
    dst0[pos] = blend_uchar (dst0[pos], sample.x, sample.w);
    dst1[pos] = blend_uchar (dst1[pos], sample.y, sample.w);
    dst2[pos] = blend_uchar (dst2[pos], sample.z, sample.w);
  }
};

struct OutputBGRP : public IOutput
{
  __device__ void
  Write (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,
      unsigned char * dst3, float4 sample, int x, int y, int stride0,
      int stride1)
  {
    int pos = x + y * stride0;
    dst0[pos] = scale_to_uchar (sample.z);
    dst1[pos] = scale_to_uchar (sample.y);
    dst2[pos] = scale_to_uchar (sample.x);
  }

  __device__ void
  Blend (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,
      unsigned char * dst3, float4 sample, int x, int y, int stride0,
      int stride1)
  {
    int pos = x + y * stride0;
    dst0[pos] = blend_uchar (dst0[pos], sample.z, sample.w);
    dst1[pos] = blend_uchar (dst1[pos], sample.y, sample.w);
    dst2[pos] = blend_uchar (dst2[pos], sample.x, sample.w);
  }
};

struct OutputGBR : public IOutput
{
  __device__ void
  Write (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,
      unsigned char * dst3, float4 sample, int x, int y, int stride0,
      int stride1)
  {
    int pos = x + y * stride0;
    dst0[pos] = scale_to_uchar (sample.y);
    dst1[pos] = scale_to_uchar (sample.z);
    dst2[pos] = scale_to_uchar (sample.x);
  }

  __device__ void
  Blend (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,
      unsigned char * dst3, float4 sample, int x, int y, int stride0,
      int stride1)
  {
    int pos = x + y * stride0;
    dst0[pos] = blend_uchar (dst0[pos], sample.y, sample.w);
    dst1[pos] = blend_uchar (dst1[pos], sample.z, sample.w);
    dst2[pos] = blend_uchar (dst2[pos], sample.x, sample.w);
  }
};

struct OutputGBR_10 : public IOutput
{
  __device__ void
  Write (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,
      unsigned char * dst3, float4 sample, int x, int y, int stride0,
      int stride1)
  {
    int pos = x * 2 + y * stride0;
    *(unsigned short *) &dst0[pos] = scale_to_10bits (sample.y);
    *(unsigned short *) &dst1[pos] = scale_to_10bits (sample.z);
    *(unsigned short *) &dst2[pos] = scale_to_10bits (sample.x);
  }

  __device__ void
  Blend (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,
      unsigned char * dst3, float4 sample, int x, int y, int stride0,
      int stride1)
  {
    int pos = x * 2 + y * stride0;
    unsigned short * target = (unsigned short *) &dst0[pos];
    *target = blend_10bits (*target, sample.y, sample.w);
    target = (unsigned short *) &dst1[pos];
    *target = blend_10bits (*target, sample.z, sample.w);
    target = (unsigned short *) &dst2[pos];
    *target = blend_10bits (*target, sample.x, sample.w);
  }
};

struct OutputGBR_12 : public IOutput
{
  __device__ void
  Write (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,
      unsigned char * dst3, float4 sample, int x, int y, int stride0,
      int stride1)
  {
    int pos = x * 2 + y * stride0;
    *(unsigned short *) &dst0[pos] = scale_to_12bits (sample.y);
    *(unsigned short *) &dst1[pos] = scale_to_12bits (sample.z);
    *(unsigned short *) &dst2[pos] = scale_to_12bits (sample.x);
  }

  __device__ void
  Blend (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,
      unsigned char * dst3, float4 sample, int x, int y, int stride0,
      int stride1)
  {
    int pos = x * 2 + y * stride0;
    unsigned short * target = (unsigned short *) &dst0[pos];
    *target = blend_12bits (*target, sample.y, sample.w);
    target = (unsigned short *) &dst1[pos];
    *target = blend_12bits (*target, sample.z, sample.w);
    target = (unsigned short *) &dst2[pos];
    *target = blend_12bits (*target, sample.x, sample.w);
  }
};

struct OutputGBR_16 : public IOutput
{
  __device__ void
  Write (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,
      unsigned char * dst3, float4 sample, int x, int y, int stride0,
      int stride1)
  {
    int pos = x * 2 + y * stride0;
    *(unsigned short *) &dst0[pos] = scale_to_ushort (sample.y);
    *(unsigned short *) &dst1[pos] = scale_to_ushort (sample.z);
    *(unsigned short *) &dst2[pos] = scale_to_ushort (sample.x);
  }

  __device__ void
  Blend (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,
      unsigned char * dst3, float4 sample, int x, int y, int stride0,
      int stride1)
  {
    int pos = x * 2 + y * stride0;
    unsigned short * target = (unsigned short *) &dst0[pos];
    *target = blend_ushort (*target, sample.y, sample.w);
    target = (unsigned short *) &dst1[pos];
    *target = blend_ushort (*target, sample.z, sample.w);
    target = (unsigned short *) &dst2[pos];
    *target = blend_ushort (*target, sample.x, sample.w);
  }
};

struct OutputGBRA : public IOutput
{
  __device__ void
  Write (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,
      unsigned char * dst3, float4 sample, int x, int y, int stride0,
      int stride1)
  {
    int pos = x + y * stride0;
    dst0[pos] = scale_to_uchar (sample.y);
    dst1[pos] = scale_to_uchar (sample.z);
    dst2[pos] = scale_to_uchar (sample.x);
    dst3[pos] = scale_to_uchar (sample.w);
  }

  __device__ void
  Blend (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,
      unsigned char * dst3, float4 sample, int x, int y, int stride0,
      int stride1)
  {
    int pos = x + y * stride0;
    dst0[pos] = blend_uchar (dst0[pos], sample.y, sample.w);
    dst1[pos] = blend_uchar (dst1[pos], sample.z, sample.w);
    dst2[pos] = blend_uchar (dst2[pos], sample.x, sample.w);
    dst3[pos] = blend_uchar (dst3[pos], 1.0, sample.w);
  }
};

struct OutputVUYA : public IOutput
{
  __device__ void
  Write (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,
      unsigned char * dst3, float4 sample, int x, int y, int stride0,
      int stride1)
  {
    int pos = x * 4 + y * stride0;
    dst0[pos] = scale_to_uchar (sample.z);
    dst0[pos + 1] = scale_to_uchar (sample.y);
    dst0[pos + 2] = scale_to_uchar (sample.x);
    dst0[pos + 3] = scale_to_uchar (sample.w);
  }

  __device__ void
  Blend (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,
      unsigned char * dst3, float4 sample, int x, int y, int stride0,
      int stride1)
  {
    int pos = x * 4 + y * stride0;
    dst0[pos] = blend_uchar (dst0[pos], sample.z, sample.w);
    dst0[pos + 1] = blend_uchar (dst0[pos + 1], sample.y, sample.w);
    dst0[pos + 2] = blend_uchar (dst0[pos + 2], sample.x, sample.w);
    dst0[pos + 3] = blend_uchar (dst0[pos + 3], 1.0, sample.w);
  }
};

__device__ inline float2
rotate_identity (float x, float y)
{
  return make_float2(x, y);
}

__device__ inline float2
rotate_90r (float x, float y)
{
  return make_float2(y, 1.0 - x);
}

__device__ inline float2
rotate_180 (float x, float y)
{
  return make_float2(1.0 - x, 1.0 - y);
}

__device__ inline float2
rotate_90l (float x, float y)
{
  return make_float2(1.0 - y, x);
}

__device__ inline float2
rotate_horiz (float x, float y)
{
  return make_float2(1.0 - x, y);
}

__device__ inline float2
rotate_vert (float x, float y)
{
  return make_float2(x, 1.0 - y);
}

__device__ inline float2
rotate_ul_lr (float x, float y)
{
  return make_float2(y, x);
}

__device__ inline float2
rotate_ur_ll (float x, float y)
{
  return make_float2(1.0 - y, 1.0 - x);
}
__device__ inline float2
do_rotate (float x, float y, int direction)
{
  switch (direction) {
    case 1:
      return rotate_90r (x, y);
    case 2:
      return rotate_180 (x, y);
    case 3:
      return rotate_90l (x, y);
    case 4:
      return rotate_horiz (x, y);
    case 5:
      return rotate_vert (x, y);
    case 6:
      return rotate_ul_lr (x, y);
    case 7:
      return rotate_ur_ll (x, y);
    default:
      return rotate_identity (x, y);
  }
}

extern "C" {
__global__ void
GstHipConverterMain (TextureObject_t tex0, TextureObject_t tex1,
    TextureObject_t tex2, TextureObject_t tex3, unsigned char * dst0,
    unsigned char * dst1, unsigned char * dst2, unsigned char * dst3,
    int stride0, int stride1, ConstBuffer const_buf, int off_x, int off_y)
{
  ConvertSimple g_converter;
  SAMPLER g_sampler;
  OUTPUT g_output;
  int x_pos = blockIdx.x * blockDim.x + threadIdx.x + off_x;
  int y_pos = blockIdx.y * blockDim.y + threadIdx.y + off_y;
  float4 sample;
  if (x_pos >= const_buf.width || y_pos >= const_buf.height ||
      const_buf.view_width <= 0 || const_buf.view_height <= 0)
    return;
  if (x_pos < const_buf.left || x_pos >= const_buf.right ||
      y_pos < const_buf.top || y_pos >= const_buf.bottom) {
    if (!const_buf.fill_border)
      return;
    sample = make_float4 (const_buf.border_x, const_buf.border_y,
       const_buf.border_z, const_buf.border_w);
  } else {
    float x = (__int2float_rz (x_pos - const_buf.left) + 0.5) / const_buf.view_width;
    if (x < 0.0 || x > 1.0)
      return;
    float y = (__int2float_rz (y_pos - const_buf.top) + 0.5) / const_buf.view_height;
    if (y < 0.0 || y > 1.0)
      return;
    float2 rotated = do_rotate (x, y, const_buf.video_direction);
    float4 s = g_sampler.Execute (tex0, tex1, tex2, tex3, rotated.x, rotated.y);
    float3 rgb = make_float3 (s.x, s.y, s.z);
    float3 yuv;
    if (const_buf.do_convert)
      yuv = g_converter.Execute (rgb, &const_buf.matrix);
    else
      yuv = rgb;
    sample = make_float4 (yuv.x, yuv.y, yuv.z, s.w);
  }
  sample.w = sample.w * const_buf.alpha;
  if (!const_buf.do_blend) {
    g_output.Write (dst0, dst1, dst2, dst3, sample, x_pos, y_pos, stride0, stride1);
  } else {
    g_output.Blend (dst0, dst1, dst2, dst3, sample, x_pos, y_pos, stride0, stride1);
  }
}
}
#else
static const char ConverterMain_str[] =
"struct ColorMatrix\n"
"{\n"
"  float CoeffX[3];\n"
"  float CoeffY[3];\n"
"  float CoeffZ[3];\n"
"  float Offset[3];\n"
"  float Min[3];\n"
"  float Max[3];\n"
"};\n"
"\n"
"struct ConstBuffer\n"
"{\n"
"  ColorMatrix matrix;\n"
"  int width;\n"
"  int height;\n"
"  int left;\n"
"  int top;\n"
"  int right;\n"
"  int bottom;\n"
"  int view_width;\n"
"  int view_height;\n"
"  float border_x;\n"
"  float border_y;\n"
"  float border_z;\n"
"  float border_w;\n"
"  int fill_border;\n"
"  int video_direction;\n"
"  float alpha;\n"
"  int do_blend;\n"
"  int do_convert;\n"
"};\n"
"\n"
"__device__ inline float\n"
"dot (const float coeff[3], float3 val)\n"
"{\n"
"  return coeff[0] * val.x + coeff[1] * val.y + coeff[2] * val.z;\n"
"}\n"
"\n"
"__device__ inline float\n"
"clamp (float val, float min_val, float max_val)\n"
"{\n"
"  return max (min_val, min (val, max_val));\n"
"}\n"
"\n"
"__device__ inline float3\n"
"clamp3 (float3 val, const float min_val[3], const float max_val[3])\n"
"{\n"
"  return make_float3 (clamp (val.x, min_val[0], max_val[0]),\n"
"      clamp (val.y, min_val[1], max_val[2]),\n"
"      clamp (val.z, min_val[1], max_val[2]));\n"
"}\n"
"\n"
"__device__ inline unsigned char\n"
"scale_to_2bits (float val)\n"
"{\n"
"  return (unsigned short) __float2int_rz (val * 3.0);\n"
"}\n"
"\n"
"__device__ inline unsigned char\n"
"scale_to_uchar (float val)\n"
"{\n"
"  return (unsigned char) __float2int_rz (val * 255.0);\n"
"}\n"
"\n"
"__device__ inline unsigned short\n"
"scale_to_ushort (float val)\n"
"{\n"
"  return (unsigned short) __float2int_rz (val * 65535.0);\n"
"}\n"
"\n"
"__device__ inline unsigned short\n"
"scale_to_10bits (float val)\n"
"{\n"
"  return (unsigned short) __float2int_rz (val * 1023.0);\n"
"}\n"
"\n"
"__device__ inline unsigned short\n"
"scale_to_12bits (float val)\n"
"{\n"
"  return (unsigned short) __float2int_rz (val * 4095.0);\n"
"}\n"
"\n"
"__device__ inline unsigned char\n"
"blend_uchar (unsigned char dst, float src, float src_alpha)\n"
"{\n"
"  // DstColor' = SrcA * SrcColor + (1 - SrcA) DstColor\n"
"  float src_val = src * src_alpha;\n"
"  float dst_val = __int2float_rz (dst) / 255.0 * (1.0 - src_alpha);\n"
"  return scale_to_uchar(clamp(src_val + dst_val, 0, 1.0));\n"
"}\n"
"\n"
"__device__ inline unsigned short\n"
"blend_ushort (unsigned short dst, float src, float src_alpha)\n"
"{\n"
"  // DstColor' = SrcA * SrcColor + (1 - SrcA) DstColor\n"
"  float src_val = src * src_alpha;\n"
"  float dst_val = __int2float_rz (dst) / 65535.0 * (1.0 - src_alpha);\n"
"  return scale_to_ushort(clamp(src_val + dst_val, 0, 1.0));\n"
"}\n"
"\n"
"__device__ inline unsigned short\n"
"blend_10bits (unsigned short dst, float src, float src_alpha)\n"
"{\n"
"  // DstColor' = SrcA * SrcColor + (1 - SrcA) DstColor\n"
"  float src_val = src * src_alpha;\n"
"  float dst_val = __int2float_rz (dst) / 1023.0 * (1.0 - src_alpha);\n"
"  return scale_to_10bits(clamp(src_val + dst_val, 0, 1.0));\n"
"}\n"
"\n"
"__device__ inline unsigned short\n"
"blend_12bits (unsigned short dst, float src, float src_alpha)\n"
"{\n"
"  // DstColor' = SrcA * SrcColor + (1 - SrcA) DstColor\n"
"  float src_val = src * src_alpha;\n"
"  float dst_val = __int2float_rz (dst) / 4095.0 * (1.0 - src_alpha);\n"
"  return scale_to_12bits(clamp(src_val + dst_val, 0, 1.0));\n"
"}\n"
"\n"
"struct IConverter\n"
"{\n"
"  __device__ virtual float3\n"
"  Execute (float3 sample, const ColorMatrix * matrix) = 0;\n"
"};\n"
"\n"
"struct ConvertSimple : public IConverter\n"
"{\n"
"  __device__ float3\n"
"  Execute (float3 sample, const ColorMatrix * matrix)\n"
"  {\n"
"    float3 out;\n"
"    out.x = dot (matrix->CoeffX, sample);\n"
"    out.y = dot (matrix->CoeffY, sample);\n"
"    out.z = dot (matrix->CoeffZ, sample);\n"
"    out.x += matrix->Offset[0];\n"
"    out.y += matrix->Offset[1];\n"
"    out.z += matrix->Offset[2];\n"
"    return clamp3 (out, matrix->Min, matrix->Max);\n"
"  }\n"
"};\n"
"\n"
"struct ISampler\n"
"{\n"
"  __device__ virtual float4\n"
"  Execute (TextureObject_t tex0, TextureObject_t tex1,\n"
"      TextureObject_t tex2, TextureObject_t tex3, float x, float y) = 0;\n"
"};\n"
"\n"
"struct SampleI420 : public ISampler\n"
"{\n"
"  __device__ float4\n"
"  Execute (TextureObject_t tex0, TextureObject_t tex1,\n"
"      TextureObject_t tex2, TextureObject_t tex3, float x, float y)\n"
"  {\n"
"    float luma = tex2D<float>(tex0, x, y);\n"
"    float u = tex2D<float>(tex1, x, y);\n"
"    float v = tex2D<float>(tex2, x, y);\n"
"    return make_float4 (luma, u, v, 1);\n"
"  }\n"
"};\n"
"\n"
"struct SampleYV12 : public ISampler\n"
"{\n"
"  __device__ float4\n"
"  Execute (TextureObject_t tex0, TextureObject_t tex1,\n"
"      TextureObject_t tex2, TextureObject_t tex3, float x, float y)\n"
"  {\n"
"    float luma = tex2D<float>(tex0, x, y);\n"
"    float u = tex2D<float>(tex2, x, y);\n"
"    float v = tex2D<float>(tex1, x, y);\n"
"    return make_float4 (luma, u, v, 1);\n"
"  }\n"
"};\n"
"\n"
"struct SampleI420_10 : public ISampler\n"
"{\n"
"  __device__ float4\n"
"  Execute (TextureObject_t tex0, TextureObject_t tex1,\n"
"      TextureObject_t tex2, TextureObject_t tex3, float x, float y)\n"
"  {\n"
"    float luma = tex2D<float>(tex0, x, y);\n"
"    float u = tex2D<float>(tex1, x, y);\n"
"    float v = tex2D<float>(tex2, x, y);\n"
"    /* (1 << 6) to scale [0, 1.0) range */\n"
"    return make_float4 (luma * 64, u * 64, v * 64, 1);\n"
"  }\n"
"};\n"
"\n"
"struct SampleI420_12 : public ISampler\n"
"{\n"
"  __device__ float4\n"
"  Execute (TextureObject_t tex0, TextureObject_t tex1,\n"
"      TextureObject_t tex2, TextureObject_t tex3, float x, float y)\n"
"  {\n"
"    float luma = tex2D<float>(tex0, x, y);\n"
"    float u = tex2D<float>(tex1, x, y);\n"
"    float v = tex2D<float>(tex2, x, y);\n"
"    /* (1 << 4) to scale [0, 1.0) range */\n"
"    return make_float4 (luma * 16, u * 16, v * 16, 1);\n"
"  }\n"
"};\n"
"\n"
"struct SampleNV12 : public ISampler\n"
"{\n"
"  __device__ float4\n"
"  Execute (TextureObject_t tex0, TextureObject_t tex1,\n"
"      TextureObject_t tex2, TextureObject_t tex3, float x, float y)\n"
"  {\n"
"    float luma = tex2D<float>(tex0, x, y);\n"
"    float2 uv = tex2D<float2>(tex1, x, y);\n"
"    return make_float4 (luma, uv.x, uv.y, 1);\n"
"  }\n"
"};\n"
"\n"
"struct SampleNV21 : public ISampler\n"
"{\n"
"  __device__ float4\n"
"  Execute (TextureObject_t tex0, TextureObject_t tex1,\n"
"      TextureObject_t tex2, TextureObject_t tex3, float x, float y)\n"
"  {\n"
"    float luma = tex2D<float>(tex0, x, y);\n"
"    float2 vu = tex2D<float2>(tex1, x, y);\n"
"    return make_float4 (luma, vu.y, vu.x, 1);\n"
"  }\n"
"};\n"
"\n"
"struct SampleRGBA : public ISampler\n"
"{\n"
"  __device__ float4\n"
"  Execute (TextureObject_t tex0, TextureObject_t tex1,\n"
"      TextureObject_t tex2, TextureObject_t tex3, float x, float y)\n"
"  {\n"
"    return tex2D<float4>(tex0, x, y);\n"
"  }\n"
"};\n"
"\n"
"struct SampleBGRA : public ISampler\n"
"{\n"
"  __device__ float4\n"
"  Execute (TextureObject_t tex0, TextureObject_t tex1,\n"
"      TextureObject_t tex2, TextureObject_t tex3, float x, float y)\n"
"  {\n"
"    float4 bgra = tex2D<float4>(tex0, x, y);\n"
"    return make_float4 (bgra.z, bgra.y, bgra.x, bgra.w);\n"
"  }\n"
"};\n"
"\n"
"struct SampleRGBx : public ISampler\n"
"{\n"
"  __device__ float4\n"
"  Execute (TextureObject_t tex0, TextureObject_t tex1,\n"
"      TextureObject_t tex2, TextureObject_t tex3, float x, float y)\n"
"  {\n"
"    float4 rgbx = tex2D<float4>(tex0, x, y);\n"
"    rgbx.w = 1;\n"
"    return rgbx;\n"
"  }\n"
"};\n"
"\n"
"struct SampleBGRx : public ISampler\n"
"{\n"
"  __device__ float4\n"
"  Execute (TextureObject_t tex0, TextureObject_t tex1,\n"
"      TextureObject_t tex2, TextureObject_t tex3, float x, float y)\n"
"  {\n"
"    float4 bgrx = tex2D<float4>(tex0, x, y);\n"
"    return make_float4 (bgrx.z, bgrx.y, bgrx.x, 1);\n"
"  }\n"
"};\n"
"\n"
"struct SampleARGB : public ISampler\n"
"{\n"
"  __device__ float4\n"
"  Execute (TextureObject_t tex0, TextureObject_t tex1,\n"
"      TextureObject_t tex2, TextureObject_t tex3, float x, float y)\n"
"  {\n"
"   float4 argb = tex2D<float4>(tex0, x, y);\n"
"   return make_float4 (argb.y, argb.z, argb.w, argb.x);\n"
"  }\n"
"};\n"
"\n"
"struct SampleABGR : public ISampler\n"
"{\n"
"  __device__ float4\n"
"  Execute (TextureObject_t tex0, TextureObject_t tex1,\n"
"      TextureObject_t tex2, TextureObject_t tex3, float x, float y)\n"
"  {\n"
"   float4 abgr = tex2D<float4>(tex0, x, y);\n"
"   return make_float4 (abgr.w, abgr.z, abgr.y, abgr.x);\n"
"  }\n"
"};\n"
"\n"
"struct SampleRGBP : public ISampler\n"
"{\n"
"  __device__ float4\n"
"  Execute (TextureObject_t tex0, TextureObject_t tex1,\n"
"      TextureObject_t tex2, TextureObject_t tex3, float x, float y)\n"
"  {\n"
"    float r = tex2D<float>(tex0, x, y);\n"
"    float g = tex2D<float>(tex1, x, y);\n"
"    float b = tex2D<float>(tex2, x, y);\n"
"    return make_float4 (r, g, b, 1);\n"
"  }\n"
"};\n"
"\n"
"struct SampleBGRP : public ISampler\n"
"{\n"
"  __device__ float4\n"
"  Execute (TextureObject_t tex0, TextureObject_t tex1,\n"
"      TextureObject_t tex2, TextureObject_t tex3, float x, float y)\n"
"  {\n"
"    float b = tex2D<float>(tex0, x, y);\n"
"    float g = tex2D<float>(tex1, x, y);\n"
"    float r = tex2D<float>(tex2, x, y);\n"
"    return make_float4 (r, g, b, 1);\n"
"  }\n"
"};\n"
"\n"
"struct SampleGBR : public ISampler\n"
"{\n"
"  __device__ float4\n"
"  Execute (TextureObject_t tex0, TextureObject_t tex1,\n"
"      TextureObject_t tex2, TextureObject_t tex3, float x, float y)\n"
"  {\n"
"    float g = tex2D<float>(tex0, x, y);\n"
"    float b = tex2D<float>(tex1, x, y);\n"
"    float r = tex2D<float>(tex2, x, y);\n"
"    return make_float4 (r, g, b, 1);\n"
"  }\n"
"};\n"
"\n"
"struct SampleGBR_10 : public ISampler\n"
"{\n"
"  __device__ float4\n"
"  Execute (TextureObject_t tex0, TextureObject_t tex1,\n"
"      TextureObject_t tex2, TextureObject_t tex3, float x, float y)\n"
"  {\n"
"    float g = tex2D<float>(tex0, x, y);\n"
"    float b = tex2D<float>(tex1, x, y);\n"
"    float r = tex2D<float>(tex2, x, y);\n"
"    /* (1 << 6) to scale [0, 1.0) range */\n"
"    return make_float4 (r * 64, g * 64, b * 64, 1);\n"
"  }\n"
"};\n"
"\n"
"struct SampleGBR_12 : public ISampler\n"
"{\n"
"  __device__ float4\n"
"  Execute (TextureObject_t tex0, TextureObject_t tex1,\n"
"      TextureObject_t tex2, TextureObject_t tex3, float x, float y)\n"
"  {\n"
"    float g = tex2D<float>(tex0, x, y);\n"
"    float b = tex2D<float>(tex1, x, y);\n"
"    float r = tex2D<float>(tex2, x, y);\n"
"    /* (1 << 4) to scale [0, 1.0) range */\n"
"    return make_float4 (r * 16, g * 16, b * 16, 1);\n"
"  }\n"
"};\n"
"\n"
"struct SampleGBRA : public ISampler\n"
"{\n"
"  __device__ float4\n"
"  Execute (TextureObject_t tex0, TextureObject_t tex1,\n"
"      TextureObject_t tex2, TextureObject_t tex3, float x, float y)\n"
"  {\n"
"    float g = tex2D<float>(tex0, x, y);\n"
"    float b = tex2D<float>(tex1, x, y);\n"
"    float r = tex2D<float>(tex2, x, y);\n"
"    float a = tex2D<float>(tex3, x, y);\n"
"    return make_float4 (r, g, b, a);\n"
"  }\n"
"};\n"
"\n"
"struct SampleVUYA : public ISampler\n"
"{\n"
"  __device__ float4\n"
"  Execute (TextureObject_t tex0, TextureObject_t tex1,\n"
"      TextureObject_t tex2, TextureObject_t tex3, float x, float y)\n"
"  {\n"
"    float4 vuya = tex2D<float4>(tex0, x, y);\n"
"    return make_float4 (vuya.z, vuya.y, vuya.x, vuya.w);\n"
"  }\n"
"};\n"
"\n"
"struct IOutput\n"
"{\n"
"  __device__ virtual void\n"
"  Write (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,\n"
"      unsigned char * dst3, float4 sample, int x, int y, int stride0,\n"
"      int stride1) = 0;\n"
"\n"
"  __device__ virtual void\n"
"  Blend (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,\n"
"        unsigned char * dst3, float4 sample, int x, int y, int stride0,\n"
"        int stride1) = 0;\n"
"};\n"
"\n"
"struct OutputI420 : public IOutput\n"
"{\n"
"  __device__ void\n"
"  Write (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,\n"
"      unsigned char * dst3, float4 sample, int x, int y, int stride0,\n"
"      int stride1)\n"
"  {\n"
"    dst0[x + y * stride0] = scale_to_uchar (sample.x);\n"
"    if (x % 2 == 0 && y % 2 == 0) {\n"
"      unsigned int pos = x / 2 + (y / 2) * stride1;\n"
"      dst1[pos] = scale_to_uchar (sample.y);\n"
"      dst2[pos] = scale_to_uchar (sample.z);\n"
"    }\n"
"  }\n"
"\n"
"  __device__ void\n"
"  Blend (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,\n"
"      unsigned char * dst3, float4 sample, int x, int y, int stride0,\n"
"      int stride1)\n"
"  {\n"
"    unsigned int pos = x + y * stride0;\n"
"    dst0[pos] = blend_uchar (dst0[pos], sample.x, sample.w);\n"
"    if (x % 2 == 0 && y % 2 == 0) {\n"
"      pos = x / 2 + (y / 2) * stride1;\n"
"      dst1[pos] = blend_uchar (dst1[pos], sample.y, sample.w);\n"
"      dst2[pos] = blend_uchar (dst2[pos], sample.z, sample.w);\n"
"    }\n"
"  }\n"
"};\n"
"\n"
"struct OutputYV12 : public IOutput\n"
"{\n"
"  __device__ void\n"
"  Write (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,\n"
"      unsigned char * dst3, float4 sample, int x, int y, int stride0,\n"
"      int stride1)\n"
"  {\n"
"    dst0[x + y * stride0] = scale_to_uchar (sample.x);\n"
"    if (x % 2 == 0 && y % 2 == 0) {\n"
"      unsigned int pos = x / 2 + (y / 2) * stride1;\n"
"      dst1[pos] = scale_to_uchar (sample.z);\n"
"      dst2[pos] = scale_to_uchar (sample.y);\n"
"    }\n"
"  }\n"
"\n"
"  __device__ void\n"
"  Blend (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,\n"
"      unsigned char * dst3, float4 sample, int x, int y, int stride0,\n"
"      int stride1)\n"
"  {\n"
"    unsigned int pos = x + y * stride0;\n"
"    dst0[pos] = blend_uchar (dst0[pos], sample.x, sample.w);\n"
"    if (x % 2 == 0 && y % 2 == 0) {\n"
"      pos = x / 2 + (y / 2) * stride1;\n"
"      dst1[pos] = blend_uchar (dst1[pos], sample.z, sample.w);\n"
"      dst2[pos] = blend_uchar (dst2[pos], sample.y, sample.w);\n"
"    }\n"
"  }\n"
"};\n"
"\n"
"struct OutputNV12 : public IOutput\n"
"{\n"
"  __device__ void\n"
"  Write (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,\n"
"      unsigned char * dst3, float4 sample, int x, int y, int stride0,\n"
"      int stride1)\n"
"  {\n"
"    dst0[x + y * stride0] = scale_to_uchar (sample.x);\n"
"    if (x % 2 == 0 && y % 2 == 0) {\n"
"      unsigned int pos = x + (y / 2) * stride1;\n"
"      dst1[pos] = scale_to_uchar (sample.y);\n"
"      dst1[pos + 1] = scale_to_uchar (sample.z);\n"
"    }\n"
"  }\n"
"\n"
"  __device__ void\n"
"  Blend (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,\n"
"      unsigned char * dst3, float4 sample, int x, int y, int stride0,\n"
"      int stride1)\n"
"  {\n"
"    unsigned int pos = x + y * stride0;\n"
"    dst0[pos] = blend_uchar (dst0[pos], sample.x, sample.w);\n"
"    if (x % 2 == 0 && y % 2 == 0) {\n"
"      pos = x + (y / 2) * stride1;\n"
"      dst1[pos] = blend_uchar (dst1[pos], sample.y, sample.w);\n"
"      dst1[pos + 1] = blend_uchar (dst1[pos + 1], sample.z, sample.w);\n"
"    }\n"
"  }\n"
"};\n"
"\n"
"struct OutputNV21 : public IOutput\n"
"{\n"
"  __device__ void\n"
"  Write (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,\n"
"      unsigned char * dst3, float4 sample, int x, int y, int stride0,\n"
"      int stride1)\n"
"  {\n"
"    dst0[x + y * stride0] = scale_to_uchar (sample.x);\n"
"    if (x % 2 == 0 && y % 2 == 0) {\n"
"      unsigned int pos = x + (y / 2) * stride1;\n"
"      dst1[pos] = scale_to_uchar (sample.z);\n"
"      dst1[pos + 1] = scale_to_uchar (sample.y);\n"
"    }\n"
"  }\n"
"\n"
"  __device__ void\n"
"  Blend (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,\n"
"      unsigned char * dst3, float4 sample, int x, int y, int stride0,\n"
"      int stride1)\n"
"  {\n"
"    unsigned int pos = x + y * stride0;\n"
"    dst0[pos] = blend_uchar (dst0[pos], sample.x, sample.w);\n"
"    if (x % 2 == 0 && y % 2 == 0) {\n"
"      pos = x + (y / 2) * stride1;\n"
"      dst1[pos] = blend_uchar (dst1[pos], sample.z, sample.w);\n"
"      dst1[pos + 1] = blend_uchar (dst1[pos + 1], sample.y, sample.w);\n"
"    }\n"
"  }\n"
"};\n"
"\n"
"struct OutputP010 : public IOutput\n"
"{\n"
"  __device__ void\n"
"  Write (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,\n"
"      unsigned char * dst3, float4 sample, int x, int y, int stride0,\n"
"      int stride1)\n"
"  {\n"
"    *(unsigned short *) &dst0[x * 2 + y * stride0] = scale_to_ushort (sample.x);\n"
"    if (x % 2 == 0 && y % 2 == 0) {\n"
"      unsigned int pos = x * 2 + (y / 2) * stride1;\n"
"      *(unsigned short *) &dst1[pos] = scale_to_ushort (sample.y);\n"
"      *(unsigned short *) &dst1[pos + 2] = scale_to_ushort (sample.z);\n"
"    }\n"
"  }\n"
"\n"
"  __device__ void\n"
"  Blend (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,\n"
"      unsigned char * dst3, float4 sample, int x, int y, int stride0,\n"
"      int stride1)\n"
"  {\n"
"    unsigned int pos = x * 2 + y * stride0;\n"
"    unsigned short * target = (unsigned short *) &dst0[pos];\n"
"    *target = blend_ushort (*target, sample.x, sample.w);\n"
"    if (x % 2 == 0 && y % 2 == 0) {\n"
"      pos = x * 2 + (y / 2) * stride1;\n"
"      target = (unsigned short *) &dst1[pos];\n"
"      *target = blend_ushort (*target, sample.y, sample.w);\n"
"      target = (unsigned short *) &dst1[pos + 2];\n"
"      *target = blend_ushort (*target, sample.z, sample.w);\n"
"    }\n"
"  }\n"
"};\n"
"\n"
"struct OutputI420_10 : public IOutput\n"
"{\n"
"  __device__ void\n"
"  Write (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,\n"
"      unsigned char * dst3, float4 sample, int x, int y, int stride0,\n"
"      int stride1)\n"
"  {\n"
"    *(unsigned short *) &dst0[x * 2 + y * stride0] = scale_to_10bits (sample.x);\n"
"    if (x % 2 == 0 && y % 2 == 0) {\n"
"      unsigned int pos = x + (y / 2) * stride1;\n"
"      *(unsigned short *) &dst1[pos] = scale_to_10bits (sample.y);\n"
"      *(unsigned short *) &dst2[pos] = scale_to_10bits (sample.z);\n"
"    }\n"
"  }\n"
"\n"
"  __device__ void\n"
"  Blend (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,\n"
"      unsigned char * dst3, float4 sample, int x, int y, int stride0,\n"
"      int stride1)\n"
"  {\n"
"    unsigned int pos = x * 2 + y * stride0;\n"
"    unsigned short * target = (unsigned short *) &dst0[pos];\n"
"    *target = blend_10bits (*target, sample.x, sample.w);\n"
"    if (x % 2 == 0 && y % 2 == 0) {\n"
"      pos = x * 2 + (y / 2) * stride1;\n"
"      target = (unsigned short *) &dst1[pos];\n"
"      *target = blend_10bits (*target, sample.y, sample.w);\n"
"      target = (unsigned short *) &dst2[pos];\n"
"      *target = blend_10bits (*target, sample.z, sample.w);\n"
"    }\n"
"  }\n"
"};\n"
"\n"
"struct OutputI420_12 : public IOutput\n"
"{\n"
"  __device__ void\n"
"  Write (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,\n"
"      unsigned char * dst3, float4 sample, int x, int y, int stride0,\n"
"      int stride1)\n"
"  {\n"
"    *(unsigned short *) &dst0[x * 2 + y * stride0] = scale_to_12bits (sample.x);\n"
"    if (x % 2 == 0 && y % 2 == 0) {\n"
"      unsigned int pos = x + (y / 2) * stride1;\n"
"      *(unsigned short *) &dst1[pos] = scale_to_12bits (sample.y);\n"
"      *(unsigned short *) &dst2[pos] = scale_to_12bits (sample.z);\n"
"    }\n"
"  }\n"
"\n"
"  __device__ void\n"
"  Blend (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,\n"
"      unsigned char * dst3, float4 sample, int x, int y, int stride0,\n"
"      int stride1)\n"
"  {\n"
"    unsigned int pos = x * 2 + y * stride0;\n"
"    unsigned short * target = (unsigned short *) &dst0[pos];\n"
"    *target = blend_12bits (*target, sample.x, sample.w);\n"
"    if (x % 2 == 0 && y % 2 == 0) {\n"
"      pos = x * 2 + (y / 2) * stride1;\n"
"      target = (unsigned short *) &dst1[pos];\n"
"      *target = blend_12bits (*target, sample.y, sample.w);\n"
"      target = (unsigned short *) &dst2[pos];\n"
"      *target = blend_12bits (*target, sample.z, sample.w);\n"
"    }\n"
"  }\n"
"};\n"
"\n"
"struct OutputY444 : public IOutput\n"
"{\n"
"  __device__ void\n"
"  Write (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,\n"
"      unsigned char * dst3, float4 sample, int x, int y, int stride0,\n"
"      int stride1)\n"
"  {\n"
"    int pos = x + y * stride0;\n"
"    dst0[pos] = scale_to_uchar (sample.x);\n"
"    dst1[pos] = scale_to_uchar (sample.y);\n"
"    dst2[pos] = scale_to_uchar (sample.z);\n"
"  }\n"
"\n"
"  __device__ void\n"
"  Blend (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,\n"
"      unsigned char * dst3, float4 sample, int x, int y, int stride0,\n"
"      int stride1)\n"
"  {\n"
"    int pos = x + y * stride0;\n"
"    dst0[pos] = blend_uchar (dst0[pos], sample.x, sample.w);\n"
"    dst1[pos] = blend_uchar (dst1[pos], sample.y, sample.w);\n"
"    dst2[pos] = blend_uchar (dst2[pos], sample.z, sample.w);\n"
"  }\n"
"};\n"
"\n"
"struct OutputY444_10 : public IOutput\n"
"{\n"
"  __device__ void\n"
"  Write (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,\n"
"      unsigned char * dst3, float4 sample, int x, int y, int stride0,\n"
"      int stride1)\n"
"  {\n"
"    int pos = x * 2 + y * stride0;\n"
"    *(unsigned short *) &dst0[pos] = scale_to_10bits (sample.x);\n"
"    *(unsigned short *) &dst1[pos] = scale_to_10bits (sample.y);\n"
"    *(unsigned short *) &dst2[pos] = scale_to_10bits (sample.z);\n"
"  }\n"
"\n"
"  __device__ void\n"
"  Blend (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,\n"
"      unsigned char * dst3, float4 sample, int x, int y, int stride0,\n"
"      int stride1)\n"
"  {\n"
"    int pos = x * 2 + y * stride0;\n"
"    unsigned short * target = (unsigned short *) &dst0[pos];\n"
"    *target = blend_10bits (*target, sample.x, sample.w);\n"
"    target = (unsigned short *) &dst1[pos];\n"
"    *target = blend_10bits (*target, sample.y, sample.w);\n"
"    target = (unsigned short *) &dst2[pos];\n"
"    *target = blend_10bits (*target, sample.z, sample.w);\n"
"  }\n"
"};\n"
"\n"
"struct OutputY444_12 : public IOutput\n"
"{\n"
"  __device__ void\n"
"  Write (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,\n"
"      unsigned char * dst3, float4 sample, int x, int y, int stride0,\n"
"      int stride1)\n"
"  {\n"
"    int pos = x * 2 + y * stride0;\n"
"    *(unsigned short *) &dst0[pos] = scale_to_12bits (sample.x);\n"
"    *(unsigned short *) &dst1[pos] = scale_to_12bits (sample.y);\n"
"    *(unsigned short *) &dst2[pos] = scale_to_12bits (sample.z);\n"
"  }\n"
"\n"
"  __device__ void\n"
"  Blend (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,\n"
"      unsigned char * dst3, float4 sample, int x, int y, int stride0,\n"
"      int stride1)\n"
"  {\n"
"    int pos = x * 2 + y * stride0;\n"
"    unsigned short * target = (unsigned short *) &dst0[pos];\n"
"    *target = blend_12bits (*target, sample.x, sample.w);\n"
"    target = (unsigned short *) &dst1[pos];\n"
"    *target = blend_12bits (*target, sample.y, sample.w);\n"
"    target = (unsigned short *) &dst2[pos];\n"
"    *target = blend_12bits (*target, sample.z, sample.w);\n"
"  }\n"
"};\n"
"\n"
"struct OutputY444_16 : public IOutput\n"
"{\n"
"  __device__ void\n"
"  Write (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,\n"
"      unsigned char * dst3, float4 sample, int x, int y, int stride0,\n"
"      int stride1)\n"
"  {\n"
"    int pos = x * 2 + y * stride0;\n"
"    *(unsigned short *) &dst0[pos] = scale_to_ushort (sample.x);\n"
"    *(unsigned short *) &dst1[pos] = scale_to_ushort (sample.y);\n"
"    *(unsigned short *) &dst2[pos] = scale_to_ushort (sample.z);\n"
"  }\n"
"\n"
"  __device__ void\n"
"  Blend (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,\n"
"      unsigned char * dst3, float4 sample, int x, int y, int stride0,\n"
"      int stride1)\n"
"  {\n"
"    int pos = x * 2 + y * stride0;\n"
"    unsigned short * target = (unsigned short *) &dst0[pos];\n"
"    *target = blend_ushort (*target, sample.x, sample.w);\n"
"    target = (unsigned short *) &dst1[pos];\n"
"    *target = blend_ushort (*target, sample.y, sample.w);\n"
"    target = (unsigned short *) &dst2[pos];\n"
"    *target = blend_ushort (*target, sample.z, sample.w);\n"
"  }\n"
"};\n"
"\n"
"struct OutputRGBA : public IOutput\n"
"{\n"
"  __device__ void\n"
"  Write (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,\n"
"      unsigned char * dst3, float4 sample, int x, int y, int stride0,\n"
"      int stride1)\n"
"  {\n"
"    int pos = x * 4 + y * stride0;\n"
"    dst0[pos] = scale_to_uchar (sample.x);\n"
"    dst0[pos + 1] = scale_to_uchar (sample.y);\n"
"    dst0[pos + 2] = scale_to_uchar (sample.z);\n"
"    dst0[pos + 3] = scale_to_uchar (sample.w);\n"
"  }\n"
"\n"
"  __device__ void\n"
"  Blend (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,\n"
"      unsigned char * dst3, float4 sample, int x, int y, int stride0,\n"
"      int stride1)\n"
"  {\n"
"    int pos = x * 4 + y * stride0;\n"
"    dst0[pos] = blend_uchar (dst0[pos], sample.x, sample.w);\n"
"    dst0[pos + 1] = blend_uchar (dst0[pos + 1], sample.y, sample.w);\n"
"    dst0[pos + 2] = blend_uchar (dst0[pos + 2], sample.z, sample.w);\n"
"    dst0[pos + 3] = blend_uchar (dst0[pos + 3], 1.0, sample.w);\n"
"  }\n"
"};\n"
"\n"
"struct OutputRGBx : public IOutput\n"
"{\n"
"  __device__ void\n"
"  Write (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,\n"
"      unsigned char * dst3, float4 sample, int x, int y, int stride0,\n"
"      int stride1)\n"
"  {\n"
"    int pos = x * 4 + y * stride0;\n"
"    dst0[pos] = scale_to_uchar (sample.x);\n"
"    dst0[pos + 1] = scale_to_uchar (sample.y);\n"
"    dst0[pos + 2] = scale_to_uchar (sample.z);\n"
"    dst0[pos + 3] = 255;\n"
"  }\n"
"\n"
"  __device__ void\n"
"  Blend (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,\n"
"      unsigned char * dst3, float4 sample, int x, int y, int stride0,\n"
"      int stride1)\n"
"  {\n"
"    int pos = x * 4 + y * stride0;\n"
"    dst0[pos] = blend_uchar (dst0[pos], sample.x, sample.w);\n"
"    dst0[pos + 1] = blend_uchar (dst0[pos + 1], sample.y, sample.w);\n"
"    dst0[pos + 2] = blend_uchar (dst0[pos + 2], sample.z, sample.w);\n"
"    dst0[pos + 3] = 255;\n"
"  }\n"
"};\n"
"\n"
"struct OutputBGRA : public IOutput\n"
"{\n"
"  __device__ void\n"
"  Write (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,\n"
"      unsigned char * dst3, float4 sample, int x, int y, int stride0,\n"
"      int stride1)\n"
"  {\n"
"    int pos = x * 4 + y * stride0;\n"
"    dst0[pos] = scale_to_uchar (sample.z);\n"
"    dst0[pos + 1] = scale_to_uchar (sample.y);\n"
"    dst0[pos + 2] = scale_to_uchar (sample.x);\n"
"    dst0[pos + 3] = scale_to_uchar (sample.w);\n"
"  }\n"
"\n"
"  __device__ void\n"
"  Blend (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,\n"
"      unsigned char * dst3, float4 sample, int x, int y, int stride0,\n"
"      int stride1)\n"
"  {\n"
"    int pos = x * 4 + y * stride0;\n"
"    dst0[pos] = blend_uchar (dst0[pos], sample.z, sample.w);\n"
"    dst0[pos + 1] = blend_uchar (dst0[pos + 1], sample.y, sample.w);\n"
"    dst0[pos + 2] = blend_uchar (dst0[pos + 2], sample.x, sample.w);\n"
"    dst0[pos + 3] = blend_uchar (dst0[pos + 3], 1.0, sample.w);\n"
"  }\n"
"};\n"
"\n"
"struct OutputBGRx : public IOutput\n"
"{\n"
"  __device__ void\n"
"  Write (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,\n"
"      unsigned char * dst3, float4 sample, int x, int y, int stride0,\n"
"      int stride1)\n"
"  {\n"
"    int pos = x * 4 + y * stride0;\n"
"    dst0[pos] = scale_to_uchar (sample.z);\n"
"    dst0[pos + 1] = scale_to_uchar (sample.y);\n"
"    dst0[pos + 2] = scale_to_uchar (sample.x);\n"
"    dst0[pos + 3] = 255;\n"
"  }\n"
"\n"
"  __device__ void\n"
"  Blend (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,\n"
"      unsigned char * dst3, float4 sample, int x, int y, int stride0,\n"
"      int stride1)\n"
"  {\n"
"    int pos = x * 4 + y * stride0;\n"
"    dst0[pos] = blend_uchar (dst0[pos], sample.z, sample.w);\n"
"    dst0[pos + 1] = blend_uchar (dst0[pos + 1], sample.y, sample.w);\n"
"    dst0[pos + 2] = blend_uchar (dst0[pos + 2], sample.x, sample.w);\n"
"    dst0[pos + 3] = 255;\n"
"  }\n"
"};\n"
"\n"
"struct OutputARGB : public IOutput\n"
"{\n"
"  __device__ void\n"
"  Write (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,\n"
"      unsigned char * dst3, float4 sample, int x, int y, int stride0,\n"
"      int stride1)\n"
"  {\n"
"    int pos = x * 4 + y * stride0;\n"
"    dst0[pos] = scale_to_uchar (sample.w);\n"
"    dst0[pos + 1] = scale_to_uchar (sample.x);\n"
"    dst0[pos + 2] = scale_to_uchar (sample.y);\n"
"    dst0[pos + 3] = scale_to_uchar (sample.z);\n"
"  }\n"
"\n"
"  __device__ void\n"
"  Blend (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,\n"
"      unsigned char * dst3, float4 sample, int x, int y, int stride0,\n"
"      int stride1)\n"
"  {\n"
"    int pos = x * 4 + y * stride0;\n"
"    dst0[pos] = blend_uchar (dst0[pos], 1.0, sample.w);\n"
"    dst0[pos + 1] = blend_uchar (dst0[pos + 1], sample.x, sample.w);\n"
"    dst0[pos + 2] = blend_uchar (dst0[pos + 2], sample.y, sample.w);\n"
"    dst0[pos + 3] = blend_uchar (dst0[pos + 3], sample.z, sample.w);\n"
"  }\n"
"};\n"
"\n"
"struct OutputABGR : public IOutput\n"
"{\n"
"  __device__ void\n"
"  Write (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,\n"
"      unsigned char * dst3, float4 sample, int x, int y, int stride0,\n"
"      int stride1)\n"
"  {\n"
"    int pos = x * 4 + y * stride0;\n"
"    dst0[pos] = scale_to_uchar (sample.w);\n"
"    dst0[pos + 1] = scale_to_uchar (sample.z);\n"
"    dst0[pos + 2] = scale_to_uchar (sample.y);\n"
"    dst0[pos + 3] = scale_to_uchar (sample.x);\n"
"  }\n"
"\n"
"  __device__ void\n"
"  Blend (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,\n"
"      unsigned char * dst3, float4 sample, int x, int y, int stride0,\n"
"      int stride1)\n"
"  {\n"
"    int pos = x * 4 + y * stride0;\n"
"    dst0[pos] = blend_uchar (dst0[pos], 1.0, sample.w);\n"
"    dst0[pos + 1] = blend_uchar (dst0[pos + 1], sample.z, sample.w);\n"
"    dst0[pos + 2] = blend_uchar (dst0[pos + 2], sample.y, sample.w);\n"
"    dst0[pos + 3] = blend_uchar (dst0[pos + 3], sample.x, sample.w);\n"
"  }\n"
"};\n"
"\n"
"struct OutputRGB : public IOutput\n"
"{\n"
"  __device__ void\n"
"  Write (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,\n"
"      unsigned char * dst3, float4 sample, int x, int y, int stride0,\n"
"      int stride1)\n"
"  {\n"
"    int pos = x * 3 + y * stride0;\n"
"    dst0[pos] = scale_to_uchar (sample.x);\n"
"    dst0[pos + 1] = scale_to_uchar (sample.y);\n"
"    dst0[pos + 2] = scale_to_uchar (sample.z);\n"
"  }\n"
"\n"
"  __device__ void\n"
"  Blend (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,\n"
"      unsigned char * dst3, float4 sample, int x, int y, int stride0,\n"
"      int stride1)\n"
"  {\n"
"    int pos = x * 3 + y * stride0;\n"
"    dst0[pos] = blend_uchar (dst0[pos], sample.x, sample.w);\n"
"    dst0[pos + 1] = blend_uchar (dst0[pos + 1], sample.y, sample.w);\n"
"    dst0[pos + 2] = blend_uchar (dst0[pos + 2], sample.z, sample.w);\n"
"  }\n"
"};\n"
"\n"
"struct OutputBGR : public IOutput\n"
"{\n"
"  __device__ void\n"
"  Write (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,\n"
"      unsigned char * dst3, float4 sample, int x, int y, int stride0,\n"
"      int stride1)\n"
"  {\n"
"    int pos = x * 3 + y * stride0;\n"
"    dst0[pos] = scale_to_uchar (sample.z);\n"
"    dst0[pos + 1] = scale_to_uchar (sample.y);\n"
"    dst0[pos + 2] = scale_to_uchar (sample.x);\n"
"  }\n"
"\n"
"  __device__ void\n"
"  Blend (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,\n"
"      unsigned char * dst3, float4 sample, int x, int y, int stride0,\n"
"      int stride1)\n"
"  {\n"
"    int pos = x * 3 + y * stride0;\n"
"    dst0[pos] = blend_uchar (dst0[pos], sample.z, sample.w);\n"
"    dst0[pos + 1] = blend_uchar (dst0[pos + 1], sample.y, sample.w);\n"
"    dst0[pos + 2] = blend_uchar (dst0[pos + 2], sample.x, sample.w);\n"
"  }\n"
"};\n"
"\n"
"__device__ inline ushort3\n"
"unpack_rgb10a2 (unsigned int val)\n"
"{\n"
"  unsigned short r, g, b;\n"
"  r = (val & 0x3ff);\n"
"  r = (r << 6) | (r >> 4);\n"
"  g = ((val >> 10) & 0x3ff);\n"
"  g = (g << 6) | (g >> 4);\n"
"  b = ((val >> 20) & 0x3ff);\n"
"  b = (b << 6) | (b >> 4);\n"
"  return make_ushort3 (r, g, b);\n"
"}\n"
"\n"
"struct OutputRGB10A2 : public IOutput\n"
"{\n"
"  __device__ void\n"
"  Write (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,\n"
"      unsigned char * dst3, float4 sample, int x, int y, int stride0,\n"
"      int stride1)\n"
"  {\n"
"    unsigned int alpha = (unsigned int) scale_to_2bits (sample.w);\n"
"    unsigned int packed_rgb = alpha << 30;\n"
"    packed_rgb |= ((unsigned int) scale_to_10bits (sample.x));\n"
"    packed_rgb |= ((unsigned int) scale_to_10bits (sample.y)) << 10;\n"
"    packed_rgb |= ((unsigned int) scale_to_10bits (sample.z)) << 20;\n"
"    *(unsigned int *) &dst0[x * 4 + y * stride0] = packed_rgb;\n"
"  }\n"
"\n"
"  __device__ void\n"
"  Blend (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,\n"
"      unsigned char * dst3, float4 sample, int x, int y, int stride0,\n"
"      int stride1)\n"
"  {\n"
"    unsigned int * target = (unsigned int *) &dst0[x * 4 + y * stride0];\n"
"    ushort3 val = unpack_rgb10a2 (*target);\n"
"    unsigned int alpha = (unsigned int) scale_to_2bits (sample.w);\n"
"    unsigned int packed_rgb = alpha << 30;\n"
"    packed_rgb |= ((unsigned int) blend_10bits (val.x, sample.x, sample.w));\n"
"    packed_rgb |= ((unsigned int) blend_10bits (val.y, sample.y, sample.w)) << 10;\n"
"    packed_rgb |= ((unsigned int) blend_10bits (val.z, sample.z, sample.w)) << 20;\n"
"    *target = packed_rgb;\n"
"  }\n"
"};\n"
"\n"
"__device__ inline ushort3\n"
"unpack_bgr10a2 (unsigned int val)\n"
"{\n"
"  unsigned short r, g, b;\n"
"  b = (val & 0x3ff);\n"
"  b = (b << 6) | (b >> 4);\n"
"  g = ((val >> 10) & 0x3ff);\n"
"  g = (g << 6) | (g >> 4);\n"
"  r = ((val >> 20) & 0x3ff);\n"
"  r = (r << 6) | (r >> 4);\n"
"  return make_ushort3 (r, g, b);\n"
"}\n"
"\n"
"struct OutputBGR10A2 : public IOutput\n"
"{\n"
"  __device__ void\n"
"  Write (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,\n"
"      unsigned char * dst3, float4 sample, int x, int y, int stride0,\n"
"      int stride1)\n"
"  {\n"
"    unsigned int alpha = (unsigned int) scale_to_2bits (sample.x);\n"
"    unsigned int packed_rgb = alpha << 30;\n"
"    packed_rgb |= ((unsigned int) scale_to_10bits (sample.x)) << 20;\n"
"    packed_rgb |= ((unsigned int) scale_to_10bits (sample.y)) << 10;\n"
"    packed_rgb |= ((unsigned int) scale_to_10bits (sample.z));\n"
"    *(unsigned int *) &dst0[x * 4 + y * stride0] = packed_rgb;\n"
"  }\n"
"\n"
"  __device__ void\n"
"  Blend (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,\n"
"      unsigned char * dst3, float4 sample, int x, int y, int stride0,\n"
"      int stride1)\n"
"  {\n"
"    unsigned int * target = (unsigned int *) &dst0[x * 4 + y * stride0];\n"
"    ushort3 val = unpack_bgr10a2 (*target);\n"
"    unsigned int alpha = (unsigned int) scale_to_2bits (sample.w);\n"
"    unsigned int packed_rgb = alpha << 30;\n"
"    packed_rgb |= ((unsigned int) blend_10bits (val.x, sample.x, sample.w)) << 20;\n"
"    packed_rgb |= ((unsigned int) blend_10bits (val.y, sample.y, sample.w)) << 10;\n"
"    packed_rgb |= ((unsigned int) blend_10bits (val.z, sample.z, sample.w));\n"
"    *target = packed_rgb;\n"
"  }\n"
"};\n"
"\n"
"struct OutputY42B : public IOutput\n"
"{\n"
"  __device__ void\n"
"  Write (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,\n"
"      unsigned char * dst3, float4 sample, int x, int y, int stride0,\n"
"      int stride1)\n"
"  {\n"
"    dst0[x + y * stride0] = scale_to_uchar (sample.x);\n"
"    if (x % 2 == 0) {\n"
"      unsigned int pos = x / 2 + y * stride1;\n"
"      dst1[pos] = scale_to_uchar (sample.y);\n"
"      dst2[pos] = scale_to_uchar (sample.z);\n"
"    }\n"
"  }\n"
"\n"
"  __device__ void\n"
"  Blend (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,\n"
"      unsigned char * dst3, float4 sample, int x, int y, int stride0,\n"
"      int stride1)\n"
"  {\n"
"    unsigned int pos = x + y * stride0;\n"
"    dst0[pos] = blend_uchar (dst0[pos], sample.x, sample.w);\n"
"    if (x % 2 == 0) {\n"
"      pos = x / 2 + y * stride1;\n"
"      dst1[pos] = blend_uchar (dst1[pos], sample.y, sample.w);\n"
"      dst2[pos] = blend_uchar (dst2[pos], sample.z, sample.w);\n"
"    }\n"
"  }\n"
"};\n"
"\n"
"struct OutputI422_10 : public IOutput\n"
"{\n"
"  __device__ void\n"
"  Write (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,\n"
"      unsigned char * dst3, float4 sample, int x, int y, int stride0,\n"
"      int stride1)\n"
"  {\n"
"    *(unsigned short *) &dst0[x * 2 + y * stride0] = scale_to_10bits (sample.x);\n"
"    if (x % 2 == 0) {\n"
"      unsigned int pos = x + y * stride1;\n"
"      *(unsigned short *) &dst1[pos] = scale_to_10bits (sample.y);\n"
"      *(unsigned short *) &dst2[pos] = scale_to_10bits (sample.z);\n"
"    }\n"
"  }\n"
"\n"
"  __device__ void\n"
"  Blend (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,\n"
"      unsigned char * dst3, float4 sample, int x, int y, int stride0,\n"
"      int stride1)\n"
"  {\n"
"    unsigned int pos = x * 2 + y * stride0;\n"
"    unsigned short * target = (unsigned short *) &dst0[pos];\n"
"    *target = blend_10bits (*target, sample.x, sample.w);\n"
"    if (x % 2 == 0) {\n"
"      pos = x / 2 + y * stride1;\n"
"      target = (unsigned short *) &dst1[pos];\n"
"      *target = blend_10bits (*target, sample.y, sample.w);\n"
"      target = (unsigned short *) &dst2[pos];\n"
"      *target = blend_10bits (*target, sample.z, sample.w);\n"
"    }\n"
"  }\n"
"};\n"
"\n"
"struct OutputI422_12 : public IOutput\n"
"{\n"
"  __device__ void\n"
"  Write (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,\n"
"      unsigned char * dst3, float4 sample, int x, int y, int stride0,\n"
"      int stride1)\n"
"  {\n"
"    *(unsigned short *) &dst0[x * 2 + y * stride0] = scale_to_12bits (sample.x);\n"
"    if (x % 2 == 0) {\n"
"      unsigned int pos = x + y * stride1;\n"
"      *(unsigned short *) &dst1[pos] = scale_to_12bits (sample.y);\n"
"      *(unsigned short *) &dst2[pos] = scale_to_12bits (sample.z);\n"
"    }\n"
"  }\n"
"\n"
"  __device__ void\n"
"  Blend (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,\n"
"      unsigned char * dst3, float4 sample, int x, int y, int stride0,\n"
"      int stride1)\n"
"  {\n"
"    unsigned int pos = x * 2 + y * stride0;\n"
"    unsigned short * target = (unsigned short *) &dst0[pos];\n"
"    *target = blend_12bits (*target, sample.x, sample.w);\n"
"    if (x % 2 == 0) {\n"
"      pos = x / 2 + y * stride1;\n"
"      target = (unsigned short *) &dst1[pos];\n"
"      *target = blend_12bits (*target, sample.y, sample.w);\n"
"      target = (unsigned short *) &dst2[pos];\n"
"      *target = blend_12bits (*target, sample.z, sample.w);\n"
"    }\n"
"  }\n"
"};\n"
"\n"
"struct OutputRGBP : public IOutput\n"
"{\n"
"  __device__ void\n"
"  Write (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,\n"
"      unsigned char * dst3, float4 sample, int x, int y, int stride0,\n"
"      int stride1)\n"
"  {\n"
"    int pos = x + y * stride0;\n"
"    dst0[pos] = scale_to_uchar (sample.x);\n"
"    dst1[pos] = scale_to_uchar (sample.y);\n"
"    dst2[pos] = scale_to_uchar (sample.z);\n"
"  }\n"
"\n"
"  __device__ void\n"
"  Blend (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,\n"
"      unsigned char * dst3, float4 sample, int x, int y, int stride0,\n"
"      int stride1)\n"
"  {\n"
"    int pos = x + y * stride0;\n"
"    dst0[pos] = blend_uchar (dst0[pos], sample.x, sample.w);\n"
"    dst1[pos] = blend_uchar (dst1[pos], sample.y, sample.w);\n"
"    dst2[pos] = blend_uchar (dst2[pos], sample.z, sample.w);\n"
"  }\n"
"};\n"
"\n"
"struct OutputBGRP : public IOutput\n"
"{\n"
"  __device__ void\n"
"  Write (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,\n"
"      unsigned char * dst3, float4 sample, int x, int y, int stride0,\n"
"      int stride1)\n"
"  {\n"
"    int pos = x + y * stride0;\n"
"    dst0[pos] = scale_to_uchar (sample.z);\n"
"    dst1[pos] = scale_to_uchar (sample.y);\n"
"    dst2[pos] = scale_to_uchar (sample.x);\n"
"  }\n"
"\n"
"  __device__ void\n"
"  Blend (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,\n"
"      unsigned char * dst3, float4 sample, int x, int y, int stride0,\n"
"      int stride1)\n"
"  {\n"
"    int pos = x + y * stride0;\n"
"    dst0[pos] = blend_uchar (dst0[pos], sample.z, sample.w);\n"
"    dst1[pos] = blend_uchar (dst1[pos], sample.y, sample.w);\n"
"    dst2[pos] = blend_uchar (dst2[pos], sample.x, sample.w);\n"
"  }\n"
"};\n"
"\n"
"struct OutputGBR : public IOutput\n"
"{\n"
"  __device__ void\n"
"  Write (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,\n"
"      unsigned char * dst3, float4 sample, int x, int y, int stride0,\n"
"      int stride1)\n"
"  {\n"
"    int pos = x + y * stride0;\n"
"    dst0[pos] = scale_to_uchar (sample.y);\n"
"    dst1[pos] = scale_to_uchar (sample.z);\n"
"    dst2[pos] = scale_to_uchar (sample.x);\n"
"  }\n"
"\n"
"  __device__ void\n"
"  Blend (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,\n"
"      unsigned char * dst3, float4 sample, int x, int y, int stride0,\n"
"      int stride1)\n"
"  {\n"
"    int pos = x + y * stride0;\n"
"    dst0[pos] = blend_uchar (dst0[pos], sample.y, sample.w);\n"
"    dst1[pos] = blend_uchar (dst1[pos], sample.z, sample.w);\n"
"    dst2[pos] = blend_uchar (dst2[pos], sample.x, sample.w);\n"
"  }\n"
"};\n"
"\n"
"struct OutputGBR_10 : public IOutput\n"
"{\n"
"  __device__ void\n"
"  Write (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,\n"
"      unsigned char * dst3, float4 sample, int x, int y, int stride0,\n"
"      int stride1)\n"
"  {\n"
"    int pos = x * 2 + y * stride0;\n"
"    *(unsigned short *) &dst0[pos] = scale_to_10bits (sample.y);\n"
"    *(unsigned short *) &dst1[pos] = scale_to_10bits (sample.z);\n"
"    *(unsigned short *) &dst2[pos] = scale_to_10bits (sample.x);\n"
"  }\n"
"\n"
"  __device__ void\n"
"  Blend (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,\n"
"      unsigned char * dst3, float4 sample, int x, int y, int stride0,\n"
"      int stride1)\n"
"  {\n"
"    int pos = x * 2 + y * stride0;\n"
"    unsigned short * target = (unsigned short *) &dst0[pos];\n"
"    *target = blend_10bits (*target, sample.y, sample.w);\n"
"    target = (unsigned short *) &dst1[pos];\n"
"    *target = blend_10bits (*target, sample.z, sample.w);\n"
"    target = (unsigned short *) &dst2[pos];\n"
"    *target = blend_10bits (*target, sample.x, sample.w);\n"
"  }\n"
"};\n"
"\n"
"struct OutputGBR_12 : public IOutput\n"
"{\n"
"  __device__ void\n"
"  Write (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,\n"
"      unsigned char * dst3, float4 sample, int x, int y, int stride0,\n"
"      int stride1)\n"
"  {\n"
"    int pos = x * 2 + y * stride0;\n"
"    *(unsigned short *) &dst0[pos] = scale_to_12bits (sample.y);\n"
"    *(unsigned short *) &dst1[pos] = scale_to_12bits (sample.z);\n"
"    *(unsigned short *) &dst2[pos] = scale_to_12bits (sample.x);\n"
"  }\n"
"\n"
"  __device__ void\n"
"  Blend (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,\n"
"      unsigned char * dst3, float4 sample, int x, int y, int stride0,\n"
"      int stride1)\n"
"  {\n"
"    int pos = x * 2 + y * stride0;\n"
"    unsigned short * target = (unsigned short *) &dst0[pos];\n"
"    *target = blend_12bits (*target, sample.y, sample.w);\n"
"    target = (unsigned short *) &dst1[pos];\n"
"    *target = blend_12bits (*target, sample.z, sample.w);\n"
"    target = (unsigned short *) &dst2[pos];\n"
"    *target = blend_12bits (*target, sample.x, sample.w);\n"
"  }\n"
"};\n"
"\n"
"struct OutputGBR_16 : public IOutput\n"
"{\n"
"  __device__ void\n"
"  Write (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,\n"
"      unsigned char * dst3, float4 sample, int x, int y, int stride0,\n"
"      int stride1)\n"
"  {\n"
"    int pos = x * 2 + y * stride0;\n"
"    *(unsigned short *) &dst0[pos] = scale_to_ushort (sample.y);\n"
"    *(unsigned short *) &dst1[pos] = scale_to_ushort (sample.z);\n"
"    *(unsigned short *) &dst2[pos] = scale_to_ushort (sample.x);\n"
"  }\n"
"\n"
"  __device__ void\n"
"  Blend (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,\n"
"      unsigned char * dst3, float4 sample, int x, int y, int stride0,\n"
"      int stride1)\n"
"  {\n"
"    int pos = x * 2 + y * stride0;\n"
"    unsigned short * target = (unsigned short *) &dst0[pos];\n"
"    *target = blend_ushort (*target, sample.y, sample.w);\n"
"    target = (unsigned short *) &dst1[pos];\n"
"    *target = blend_ushort (*target, sample.z, sample.w);\n"
"    target = (unsigned short *) &dst2[pos];\n"
"    *target = blend_ushort (*target, sample.x, sample.w);\n"
"  }\n"
"};\n"
"\n"
"struct OutputGBRA : public IOutput\n"
"{\n"
"  __device__ void\n"
"  Write (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,\n"
"      unsigned char * dst3, float4 sample, int x, int y, int stride0,\n"
"      int stride1)\n"
"  {\n"
"    int pos = x + y * stride0;\n"
"    dst0[pos] = scale_to_uchar (sample.y);\n"
"    dst1[pos] = scale_to_uchar (sample.z);\n"
"    dst2[pos] = scale_to_uchar (sample.x);\n"
"    dst3[pos] = scale_to_uchar (sample.w);\n"
"  }\n"
"\n"
"  __device__ void\n"
"  Blend (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,\n"
"      unsigned char * dst3, float4 sample, int x, int y, int stride0,\n"
"      int stride1)\n"
"  {\n"
"    int pos = x + y * stride0;\n"
"    dst0[pos] = blend_uchar (dst0[pos], sample.y, sample.w);\n"
"    dst1[pos] = blend_uchar (dst1[pos], sample.z, sample.w);\n"
"    dst2[pos] = blend_uchar (dst2[pos], sample.x, sample.w);\n"
"    dst3[pos] = blend_uchar (dst3[pos], 1.0, sample.w);\n"
"  }\n"
"};\n"
"\n"
"struct OutputVUYA : public IOutput\n"
"{\n"
"  __device__ void\n"
"  Write (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,\n"
"      unsigned char * dst3, float4 sample, int x, int y, int stride0,\n"
"      int stride1)\n"
"  {\n"
"    int pos = x * 4 + y * stride0;\n"
"    dst0[pos] = scale_to_uchar (sample.z);\n"
"    dst0[pos + 1] = scale_to_uchar (sample.y);\n"
"    dst0[pos + 2] = scale_to_uchar (sample.x);\n"
"    dst0[pos + 3] = scale_to_uchar (sample.w);\n"
"  }\n"
"\n"
"  __device__ void\n"
"  Blend (unsigned char * dst0, unsigned char * dst1, unsigned char * dst2,\n"
"      unsigned char * dst3, float4 sample, int x, int y, int stride0,\n"
"      int stride1)\n"
"  {\n"
"    int pos = x * 4 + y * stride0;\n"
"    dst0[pos] = blend_uchar (dst0[pos], sample.z, sample.w);\n"
"    dst0[pos + 1] = blend_uchar (dst0[pos + 1], sample.y, sample.w);\n"
"    dst0[pos + 2] = blend_uchar (dst0[pos + 2], sample.x, sample.w);\n"
"    dst0[pos + 3] = blend_uchar (dst0[pos + 3], 1.0, sample.w);\n"
"  }\n"
"};\n"
"\n"
"__device__ inline float2\n"
"rotate_identity (float x, float y)\n"
"{\n"
"  return make_float2(x, y);\n"
"}\n"
"\n"
"__device__ inline float2\n"
"rotate_90r (float x, float y)\n"
"{\n"
"  return make_float2(y, 1.0 - x);\n"
"}\n"
"\n"
"__device__ inline float2\n"
"rotate_180 (float x, float y)\n"
"{\n"
"  return make_float2(1.0 - x, 1.0 - y);\n"
"}\n"
"\n"
"__device__ inline float2\n"
"rotate_90l (float x, float y)\n"
"{\n"
"  return make_float2(1.0 - y, x);\n"
"}\n"
"\n"
"__device__ inline float2\n"
"rotate_horiz (float x, float y)\n"
"{\n"
"  return make_float2(1.0 - x, y);\n"
"}\n"
"\n"
"__device__ inline float2\n"
"rotate_vert (float x, float y)\n"
"{\n"
"  return make_float2(x, 1.0 - y);\n"
"}\n"
"\n"
"__device__ inline float2\n"
"rotate_ul_lr (float x, float y)\n"
"{\n"
"  return make_float2(y, x);\n"
"}\n"
"\n"
"__device__ inline float2\n"
"rotate_ur_ll (float x, float y)\n"
"{\n"
"  return make_float2(1.0 - y, 1.0 - x);\n"
"}\n"
"__device__ inline float2\n"
"do_rotate (float x, float y, int direction)\n"
"{\n"
"  switch (direction) {\n"
"    case 1:\n"
"      return rotate_90r (x, y);\n"
"    case 2:\n"
"      return rotate_180 (x, y);\n"
"    case 3:\n"
"      return rotate_90l (x, y);\n"
"    case 4:\n"
"      return rotate_horiz (x, y);\n"
"    case 5:\n"
"      return rotate_vert (x, y);\n"
"    case 6:\n"
"      return rotate_ul_lr (x, y);\n"
"    case 7:\n"
"      return rotate_ur_ll (x, y);\n"
"    default:\n"
"      return rotate_identity (x, y);\n"
"  }\n"
"}\n"
"\n"
"extern \"C\" {\n"
"__global__ void\n"
"GstHipConverterMain (TextureObject_t tex0, TextureObject_t tex1,\n"
"    TextureObject_t tex2, TextureObject_t tex3, unsigned char * dst0,\n"
"    unsigned char * dst1, unsigned char * dst2, unsigned char * dst3,\n"
"    int stride0, int stride1, ConstBuffer const_buf, int off_x, int off_y)\n"
"{\n"
"  ConvertSimple g_converter;\n"
"  SAMPLER g_sampler;\n"
"  OUTPUT g_output;\n"
"  int x_pos = blockIdx.x * blockDim.x + threadIdx.x + off_x;\n"
"  int y_pos = blockIdx.y * blockDim.y + threadIdx.y + off_y;\n"
"  float4 sample;\n"
"  if (x_pos >= const_buf.width || y_pos >= const_buf.height ||\n"
"      const_buf.view_width <= 0 || const_buf.view_height <= 0)\n"
"    return;\n"
"  if (x_pos < const_buf.left || x_pos >= const_buf.right ||\n"
"      y_pos < const_buf.top || y_pos >= const_buf.bottom) {\n"
"    if (!const_buf.fill_border)\n"
"      return;\n"
"    sample = make_float4 (const_buf.border_x, const_buf.border_y,\n"
"       const_buf.border_z, const_buf.border_w);\n"
"  } else {\n"
"    float x = (__int2float_rz (x_pos - const_buf.left) + 0.5) / const_buf.view_width;\n"
"    if (x < 0.0 || x > 1.0)\n"
"      return;\n"
"    float y = (__int2float_rz (y_pos - const_buf.top) + 0.5) / const_buf.view_height;\n"
"    if (y < 0.0 || y > 1.0)\n"
"      return;\n"
"    float2 rotated = do_rotate (x, y, const_buf.video_direction);\n"
"    float4 s = g_sampler.Execute (tex0, tex1, tex2, tex3, rotated.x, rotated.y);\n"
"    float3 rgb = make_float3 (s.x, s.y, s.z);\n"
"    float3 yuv;\n"
"    if (const_buf.do_convert)\n"
"      yuv = g_converter.Execute (rgb, &const_buf.matrix);\n"
"    else\n"
"      yuv = rgb;\n"
"    sample = make_float4 (yuv.x, yuv.y, yuv.z, s.w);\n"
"  }\n"
"  sample.w = sample.w * const_buf.alpha;\n"
"  if (!const_buf.do_blend) {\n"
"    g_output.Write (dst0, dst1, dst2, dst3, sample, x_pos, y_pos, stride0, stride1);\n"
"  } else {\n"
"    g_output.Blend (dst0, dst1, dst2, dst3, sample, x_pos, y_pos, stride0, stride1);\n"
"  }\n"
"}\n"
"}\n"
"\n";
#endif